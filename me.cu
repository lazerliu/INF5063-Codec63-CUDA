#include "hip/hip_runtime.h"
extern "C" {
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
}

#include "dsp.h"
#include "me.h"


__device__
static void min_warp_reduce(int i, volatile int* values)
{
	values[i] = min(values[i], values[i + 32]);
	values[i] = min(values[i], values[i + 16]);
	values[i] = min(values[i], values[i + 8]);
	values[i] = min(values[i], values[i + 4]);
	values[i] = min(values[i], values[i + 2]);
	values[i] = min(values[i], values[i + 1]);
}

template<int block_size>
__device__
static void min_reduce(int i, int* values)
{
	if (i < block_size/2)
	{
		// Intentionally no break between cases
		switch (block_size) {
			case 1024:
				values[i] = min(values[i], values[i + 512]);
				__syncthreads();
			case 512:
				values[i] = min(values[i], values[i + 256]);
				__syncthreads();
			case 256:
				values[i] = min(values[i], values[i + 128]);
				__syncthreads();
			case 128:
				values[i] = min(values[i], values[i + 64]);
				__syncthreads();
		}

		if (i < 32)
		{
			min_warp_reduce(i, values);
		}
	}
	else
	{
		switch (block_size) {
			case 1024:
				__syncthreads();
			case 512:
				__syncthreads();
			case 256:
				__syncthreads();
			case 128:
				__syncthreads();
		}
	}
}

template<int block_size>
__device__
static void first_min_occurrence(int i, int* values, int value, int* result)
{
	min_reduce<block_size>(i, values);

	if (i == 0) {
		*result = INT_MAX;
	}

	__syncthreads();

	if (value == values[0]) {
		atomicMin(result, i);
	}
}

template<int range>
__global__
static void me_block_8x8_gpu(struct macroblock* mbs, uint8_t* orig, uint8_t* ref, int* lefts, int* rights, int* tops, int* bottoms, int w, int* vector_x, int* vector_y, int* use_mv)
{
	const int i = threadIdx.x;
	const int j = threadIdx.y;
	const int ref_mb_id = j*blockDim.x + i;

	const int mb_x = blockIdx.x;
	const int mb_y = blockIdx.y;
	const int orig_mb_id = mb_y*gridDim.x + mb_x;

	const int left = lefts[mb_x];
	const int top = tops[mb_y];
	const int right = rights[mb_x];
	const int bottom = bottoms[mb_y];

	const int mx = mb_x * 8;
	const int my = mb_y * 8;

	uint8_t* orig_block = orig + my * w + mx;
	uint8_t* ref_search_range = ref + top*w + left;

	__shared__ uint8_t shared_orig_block[64];

	if (i < 8 && j < 8)
	{
		shared_orig_block[j*8 + i] = orig_block[j*w + i];
	}

	__syncthreads();

	int block_sad = INT_MAX;

	const int range_width = right - left;
	const int range_height = bottom - top;

	const int shifts = (i % 4) * 8;

	// (i/4)*4 rounds i down to the nearest integer divisible by 4
	uint8_t* ref_block_top_row_aligned = ref_search_range + j*w + (i/4)*4;

	if (j < range_height && i < range_width)
	{
		block_sad = 0;

		for (int y = 0; y < 8; ++y)
		{
			uint32_t* ref_block_row_aligned = (uint32_t*) (ref_block_top_row_aligned + y*w);
			uint32_t ref_row_left = (ref_block_row_aligned[0] >> shifts) | (ref_block_row_aligned[1] << 32-shifts);
			uint32_t ref_row_right = (ref_block_row_aligned[1] >> shifts) | (ref_block_row_aligned[2] << 32-shifts);

			uint8_t* orig_block_row = shared_orig_block + y*8;
			uint32_t orig_row_left = *((uint32_t*) orig_block_row);
			uint32_t orig_row_right = *((uint32_t*) orig_block_row + 1);

			block_sad += __vsadu4(ref_row_left, orig_row_left);
			block_sad += __vsadu4(ref_row_right, orig_row_right);
		}
	}

	const int max_range_width = range * 2;
	const int max_range_height = range * 2;
	const int max_mb_count = max_range_width * max_range_height;

	__shared__ int block_sads[max_mb_count];

	block_sads[ref_mb_id] = block_sad;

	__syncthreads();

	__shared__ int index_result;
	first_min_occurrence<max_mb_count>(ref_mb_id, block_sads, block_sad, &index_result);

	__syncthreads();

	if (ref_mb_id == 0)
	{
		/* Here, there should be a threshold on SAD that checks if the motion vector
		     is cheaper than intraprediction. We always assume MV to be beneficial */
		struct macroblock* mb = &mbs[orig_mb_id];
		mb->use_mv = 1;
		mb->mv_x = left + (index_result % max_range_width) - mx;
		mb->mv_y = top + (index_result / max_range_width) - my;
	}
}

void c63_motion_estimate(struct c63_common *cm)
{
	struct macroblock** mbs = cm->curframe->mbs_gpu;
	yuv_t* orig = cm->curframe->orig_gpu;
	yuv_t* ref = cm->refframe->recons_gpu;

	const int frame_size_Y = cm->padw[Y_COMPONENT] * cm->padh[Y_COMPONENT] * sizeof(uint8_t);
	const int frame_size_U = cm->padw[U_COMPONENT] * cm->padh[U_COMPONENT] * sizeof(uint8_t);
	const int frame_size_V = cm->padw[V_COMPONENT] * cm->padh[V_COMPONENT] * sizeof(uint8_t);

	hipMemcpy(ref->Y, cm->refframe->recons->Y, frame_size_Y, hipMemcpyHostToDevice);
	hipMemcpy(ref->U, cm->refframe->recons->U, frame_size_U, hipMemcpyHostToDevice);
	hipMemcpy(ref->V, cm->refframe->recons->V, frame_size_V, hipMemcpyHostToDevice);

	const int wY = cm->padw[Y_COMPONENT];
	const int wU = cm->padw[U_COMPONENT];
	const int wV = cm->padw[V_COMPONENT];

	/* Luma */
	dim3 numBlocksY(cm->mb_colsY, cm->mb_rowsY);
	dim3 threadsPerBlockY(ME_RANGE_Y*2, ME_RANGE_Y*2);
	me_block_8x8_gpu<ME_RANGE_Y><<<numBlocksY, threadsPerBlockY>>>(mbs[Y_COMPONENT], orig->Y, ref->Y, cm->cuda_me.leftsY_gpu, cm->cuda_me.rightsY_gpu, cm->cuda_me.topsY_gpu, cm->cuda_me.bottomsY_gpu, wY, cm->cuda_me.vector_xY_gpu, cm->cuda_me.vector_yY_gpu, cm->cuda_me.use_mvY_gpu);
	hipMemcpy(cm->curframe->mbs[Y_COMPONENT], mbs[Y_COMPONENT], cm->mb_rowsY * cm->mb_colsY * sizeof(struct macroblock), hipMemcpyDeviceToHost);

	/* Chroma */
	dim3 numBlocksUV(cm->mb_colsUV, cm->mb_rowsUV);
	dim3 threadsPerBlockUV(ME_RANGE_UV*2, ME_RANGE_UV*2);
	me_block_8x8_gpu<ME_RANGE_UV><<<numBlocksUV, threadsPerBlockUV>>>(mbs[U_COMPONENT], orig->U, ref->U, cm->cuda_me.leftsUV_gpu, cm->cuda_me.rightsUV_gpu, cm->cuda_me.topsUV_gpu, cm->cuda_me.bottomsUV_gpu, wU, cm->cuda_me.vector_xU_gpu, cm->cuda_me.vector_yU_gpu, cm->cuda_me.use_mvU_gpu);
	hipMemcpy(cm->curframe->mbs[U_COMPONENT], mbs[U_COMPONENT], cm->mb_rowsUV * cm->mb_colsUV * sizeof(struct macroblock), hipMemcpyDeviceToHost);

	me_block_8x8_gpu<ME_RANGE_UV><<<numBlocksUV, threadsPerBlockUV>>>(mbs[V_COMPONENT], orig->V, ref->V, cm->cuda_me.leftsUV_gpu, cm->cuda_me.rightsUV_gpu, cm->cuda_me.topsUV_gpu, cm->cuda_me.bottomsUV_gpu, wV, cm->cuda_me.vector_xV_gpu, cm->cuda_me.vector_yV_gpu, cm->cuda_me.use_mvV_gpu);
	hipMemcpy(cm->curframe->mbs[V_COMPONENT], mbs[V_COMPONENT], cm->mb_rowsUV * cm->mb_colsUV * sizeof(struct macroblock), hipMemcpyDeviceToHost);
}

/* Motion compensation for 8x8 block */
__global__
static void mc_block_8x8_gpu(struct macroblock* mbs, int w, uint8_t *predicted, uint8_t *ref, int* vector_x, int* vector_y, int* use_mv)
{
	int mb_x = blockIdx.x;
	int mb_y = threadIdx.x;

	int index = mb_y * w / 8 + mb_x;

	struct macroblock* mb = &mbs[index];

	if (!mb->use_mv) {
		return;
	}

	int mv_x = mb->mv_x;
	int mv_y = mb->mv_y;

	int left = mb_x * 8;
	int top = mb_y * 8;
	int right = left + 8;
	int bottom = top + 8;

	/* Copy block from ref mandated by MV */
	int x, y;

	for (y = top; y < bottom; ++y)
	{
		for (x = left; x < right; ++x)
		{
			predicted[y * w + x] = ref[(y + mv_y) * w + (x + mv_x)];
		}
	}
}

void c63_motion_compensate(struct c63_common *cm)
{
	struct macroblock** mbs = cm->curframe->mbs_gpu;
	yuv_t* ref = cm->refframe->recons_gpu;

	int wY = cm->padw[Y_COMPONENT];
	int wU = cm->padw[U_COMPONENT];
	int wV = cm->padw[V_COMPONENT];

	/* Luma */
	// TODO: Number of macroblock rows are now limited to 1024. Number of threads per block should
	// ideally be a multiplum of the warp size (32).
	mc_block_8x8_gpu<<<cm->mb_colsY, cm->mb_rowsY>>>(mbs[Y_COMPONENT], wY, cm->cuda_me.predY_gpu, ref->Y, cm->cuda_me.vector_xY_gpu, cm->cuda_me.vector_yY_gpu, cm->cuda_me.use_mvY_gpu);
	hipMemcpy(cm->curframe->predicted->Y, cm->cuda_me.predY_gpu, cm->ypw * cm->yph * sizeof(uint8_t), hipMemcpyDeviceToHost);

	/* Chroma */
	mc_block_8x8_gpu<<<cm->mb_colsUV, cm->mb_rowsUV>>>(mbs[U_COMPONENT], wU, cm->cuda_me.predU_gpu, ref->U, cm->cuda_me.vector_xU_gpu, cm->cuda_me.vector_yU_gpu, cm->cuda_me.use_mvU_gpu);
	hipMemcpy(cm->curframe->predicted->U, cm->cuda_me.predU_gpu, cm->upw * cm->uph * sizeof(uint8_t), hipMemcpyDeviceToHost);

	mc_block_8x8_gpu<<<cm->mb_colsUV, cm->mb_rowsUV>>>(mbs[V_COMPONENT], wV, cm->cuda_me.predV_gpu, ref->V, cm->cuda_me.vector_xV_gpu, cm->cuda_me.vector_yV_gpu, cm->cuda_me.use_mvV_gpu);
	hipMemcpy(cm->curframe->predicted->V, cm->cuda_me.predV_gpu, cm->vpw * cm->vph * sizeof(uint8_t), hipMemcpyDeviceToHost);
}
