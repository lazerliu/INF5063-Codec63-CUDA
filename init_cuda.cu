#include "init_cuda.h"

struct c63_cuda init_c63_cuda()
{
	struct c63_cuda result;

	for (int c = 0; c < COLOR_COMPONENTS; ++c)
	{
		hipStreamCreate(&result.stream[c]);
		hipStreamCreate(&result.memcpy_stream[c]);

		hipEventCreate(&result.me_done[c]);
		hipEventCreate(&result.dctquant_done[c]);
	}

	return result;
}

void cleanup_c63_cuda(struct c63_cuda& c63_cuda)
{
	for (int c = 0; c < COLOR_COMPONENTS; ++c)
	{
		hipStreamDestroy(c63_cuda.stream[c]);
		hipStreamDestroy(c63_cuda.memcpy_stream[c]);

		hipEventDestroy(c63_cuda.me_done[c]);
		hipEventDestroy(c63_cuda.dctquant_done[c]);
	}
}

static struct boundaries init_me_boundaries_gpu(const struct boundaries& indata, int cols, int rows,
		hipStream_t stream)
{
	struct boundaries result;

	hipMalloc((void**) &result.left, cols * sizeof(int));
	hipMalloc((void**) &result.right, cols * sizeof(int));
	hipMalloc((void**) &result.top, rows * sizeof(int));
	hipMalloc((void**) &result.bottom, rows * sizeof(int));

	hipMemcpyAsync((void*) result.left, indata.left, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.right, indata.right, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.top, indata.top, rows * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.bottom, indata.bottom, rows * sizeof(int),
			hipMemcpyHostToDevice, stream);

	return result;
}

static void cleanup_me_boundaries_gpu(struct boundaries& boundaries_gpu)
{
	hipFree((void*) boundaries_gpu.left);
	hipFree((void*) boundaries_gpu.right);
	hipFree((void*) boundaries_gpu.top);
	hipFree((void*) boundaries_gpu.bottom);
}

struct c63_common_gpu init_c63_gpu(const struct c63_common* cm, const struct c63_cuda& c63_cuda)
{
	struct c63_common_gpu result;

	for (int c = 0; c < COLOR_COMPONENTS; ++c)
	{
		int cols = cm->mb_cols[c];
		int rows = cm->mb_rows[c];
		const struct boundaries& boundaries = cm->me_boundaries[c];
		hipStream_t stream = c63_cuda.stream[c];

		result.me_boundaries[c] = init_me_boundaries_gpu(boundaries, cols, rows, stream);
		hipMalloc(&result.sad_index_results[c], cols * rows * sizeof(unsigned int));
	}

	return result;
}

void cleanup_c63_gpu(struct c63_common_gpu& cm_gpu)
{
	for (int c = 0; c < COLOR_COMPONENTS; ++c)
	{
		cleanup_me_boundaries_gpu(cm_gpu.me_boundaries[c]);
		hipFree(cm_gpu.sad_index_results[c]);
	}
}
