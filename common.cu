#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include "dsp.h"

static const int Y = Y_COMPONENT;
static const int U = U_COMPONENT;
static const int V = V_COMPONENT;

static void init_frame_gpu(struct c63_common* cm, struct frame* f)
{
	f->recons_gpu = create_image_gpu(cm);
	f->predicted_gpu = create_image_gpu(cm);

	f->residuals_gpu = (dct_t*) malloc(sizeof(dct_t));
	hipMalloc((void**) &f->residuals_gpu->Ydct, cm->ypw * cm->yph * sizeof(int16_t));
	hipMalloc((void**) &f->residuals_gpu->Udct, cm->upw * cm->uph * sizeof(int16_t));
	hipMalloc((void**) &f->residuals_gpu->Vdct, cm->vpw * cm->vph * sizeof(int16_t));

	hipMalloc((void**) &f->mbs_gpu[Y_COMPONENT],
			cm->mb_rows[Y] * cm->mb_cols[Y] * sizeof(struct macroblock));
	hipMalloc((void**) &f->mbs_gpu[U_COMPONENT],
			cm->mb_rows[U] * cm->mb_cols[U] * sizeof(struct macroblock));
	hipMalloc((void**) &f->mbs_gpu[V_COMPONENT],
			cm->mb_rows[U] * cm->mb_cols[U] * sizeof(struct macroblock));
}

static void deinit_frame_gpu(struct frame* f)
{
	destroy_image_gpu(f->recons_gpu);
	destroy_image_gpu(f->predicted_gpu);

	hipFree(f->residuals_gpu->Ydct);
	hipFree(f->residuals_gpu->Udct);
	hipFree(f->residuals_gpu->Vdct);
	free(f->residuals_gpu);

	hipFree(f->mbs_gpu[Y_COMPONENT]);
	hipFree(f->mbs_gpu[U_COMPONENT]);
	hipFree(f->mbs_gpu[V_COMPONENT]);
}

struct macroblock *create_mb(struct macroblock *mb, size_t size, const hipStream_t& stream)
{
	hipHostMalloc((void**) &mb, size);
	hipMemsetAsync(mb, 0, size, stream);

	return mb;
}

struct frame* create_frame(struct c63_common *cm, const struct c63_cuda& c63_cuda)
{
	struct frame *f = (frame*) malloc(sizeof(struct frame));

	f->residuals = (dct_t*) malloc(sizeof(dct_t));
	hipHostMalloc((void**) &f->residuals->Ydct, cm->ypw * cm->yph * sizeof(int16_t));
	hipHostMalloc((void**) &f->residuals->Udct, cm->upw * cm->uph * sizeof(int16_t));
	hipHostMalloc((void**) &f->residuals->Vdct, cm->vpw * cm->vph * sizeof(int16_t));

	size_t sizeY = cm->mb_rows[Y] * cm->mb_cols[Y] * sizeof(struct macroblock);
	size_t sizeUV = cm->mb_rows[U] * cm->mb_cols[U] * sizeof(struct macroblock);
	f->mbs[Y_COMPONENT] = create_mb(f->mbs[Y_COMPONENT], sizeY, c63_cuda.stream[Y]);
	f->mbs[U_COMPONENT] = create_mb(f->mbs[U_COMPONENT], sizeUV, c63_cuda.stream[U]);
	f->mbs[V_COMPONENT] = create_mb(f->mbs[V_COMPONENT], sizeUV, c63_cuda.stream[V]);

	init_frame_gpu(cm, f);

	return f;
}

void destroy_frame(struct frame *f)
{
	deinit_frame_gpu(f);

	hipHostFree(f->residuals->Ydct);
	hipHostFree(f->residuals->Udct);
	hipHostFree(f->residuals->Vdct);
	free(f->residuals);

	hipHostFree(f->mbs[Y_COMPONENT]);
	hipHostFree(f->mbs[U_COMPONENT]);
	hipHostFree(f->mbs[V_COMPONENT]);

	free(f);
}

yuv_t* create_image(struct c63_common *cm)
{
	yuv_t* image = (yuv_t*) malloc(sizeof(yuv_t));
	hipHostAlloc((void**) &image->Y, cm->ypw * cm->yph * sizeof(uint8_t),
			hipHostMallocWriteCombined);
	hipHostAlloc((void**) &image->U, cm->upw * cm->uph * sizeof(uint8_t),
			hipHostMallocWriteCombined);
	hipHostAlloc((void**) &image->V, cm->vpw * cm->vph * sizeof(uint8_t),
			hipHostMallocWriteCombined);

	return image;
}

void destroy_image(yuv_t *image)
{
	hipHostFree(image->Y);
	hipHostFree(image->U);
	hipHostFree(image->V);
	free(image);
}

yuv_t* create_image_gpu(struct c63_common *cm)
{
	yuv_t* image = (yuv_t*) malloc(sizeof(yuv_t));
	hipMalloc((void**) &image->Y, cm->ypw * cm->yph * sizeof(uint8_t));
	hipMalloc((void**) &image->U, cm->upw * cm->uph * sizeof(uint8_t));
	hipMalloc((void**) &image->V, cm->vpw * cm->vph * sizeof(uint8_t));

	return image;
}

void destroy_image_gpu(yuv_t* image)
{
	hipFree(image->Y);
	hipFree(image->U);
	hipFree(image->V);
	free(image);
}

void dump_image(yuv_t *image, int w, int h, FILE *fp)
{
	fwrite(image->Y, 1, w * h, fp);
	fwrite(image->U, 1, w * h / 4, fp);
	fwrite(image->V, 1, w * h / 4, fp);
}
