#include "hip/hip_runtime.h"
extern "C" {
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
}

#include "common.h"
#include "dsp.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int16_t *gpu_frame1_16;
int16_t *gpu_frame2_16;
int16_t *gpu_out_16;

uint8_t *gpu_out_8;
uint8_t *gpu_in_8;
uint8_t *gpu_prediction_8;

void cuda_init(int width, int height){
	hipMalloc(&gpu_frame1_16, width*height*sizeof(int16_t));
	hipMalloc(&gpu_frame2_16, width*height*sizeof(int16_t));
	hipMalloc(&gpu_out_16, width*height*sizeof(int16_t));

	hipMalloc(&gpu_out_8, width*height*sizeof(uint8_t));
	hipMalloc(&gpu_in_8, width*height*sizeof(uint8_t));
	hipMalloc(&gpu_prediction_8, width*height*sizeof(uint8_t));
}

void cuda_cleanup() {
	hipFree(gpu_frame1_16);
	hipFree(gpu_frame2_16);
	hipFree(gpu_out_16);

	hipFree(gpu_in_8);
	hipFree(gpu_out_8);
	hipFree(gpu_prediction_8);
}

__global__ void dequantize_add_pred(int16_t *in_data, uint8_t *prediction, int w, int16_t *out_data) {

	int block_offset = blockIdx.x * 64;
	int x_offset = blockIdx.x * 8;

	int i = threadIdx.y;
	int j = threadIdx.x;

	/* Add prediction block. Note: DCT is not precise -
	Clamp to legal values */
	out_data[i * w + j + x_offset] = in_data[block_offset + i * 8 + j] + (int16_t) prediction[i * w + j + x_offset];
}

__global__ void dequantize_idct_row(int16_t *in_data, uint8_t *prediction, int w, int h, int y,
		int16_t *out_data, int quantization)
{
	int block_offset = blockIdx.x * 64;
 	int x_offset = blockIdx.x * 8;

 	int i = threadIdx.y;
 	int j = threadIdx.x;

	dequant_idct_block_8x8(in_data, out_data, quantization, block_offset, i, j);
	//__syncthreads();
	//out_data[i * w + j + x_offset] = temp[block_offset + i * 8 + j] + (int16_t) prediction[i * w + j + x_offset];
	//__syncthreads();
	//dequantize_end<<<numBlocks, threadsPerBlock>>>(gpu_frame2_16 + y * w, prediction, w, out_data);
	/*
	int16_t block[w*8];
	hipMemcpy(block, gpu_frame2_16 + y * w, w*8*sizeof(int16_t), hipMemcpyDeviceToHost);

	// Perform the dequantization and iDCT
	int x;
	for (x = 0; x < w; x += 8)
	{
		int i, j;

		for (i = 0; i < 8; ++i)
		{
			for (j = 0; j < 8; ++j)
			{
				// Add prediction block. Note: DCT is not precise -
				// Clamp to legal values
				int16_t tmp = block[i * 8 + j + x * 8] + (int16_t) prediction[i * w + j + x];

				if (tmp < 0)
				{
					tmp = 0;
				}
				else if (tmp > 255)
				{
					tmp = 255;
				}

				out_data[i * w + j + x] = tmp;
			}
		}
	}*/
}

void dequantize_idct(int16_t *in_data, uint8_t *prediction, uint32_t width, uint32_t height,
		uint8_t *out_data, int quantization)
{
	int numBlocks = width/8;
	dim3 threadsPerBlock(8, 8);

	unsigned int y;
	for (y = 0; y < height; y += 8)
	{
		dequantize_idct_row<<<numBlocks, threadsPerBlock>>>(in_data + y * width, prediction + y * width, width, height, y,
				gpu_frame1_16 + y * width, quantization);
		dequantize_add_pred<<<numBlocks, threadsPerBlock>>>(gpu_frame1_16 + y * width, prediction + y * width, width, gpu_out_16 + y * width);
	}

	int16_t temp[width*height];
	hipMemcpy(temp, gpu_out_16, width*height*sizeof(int16_t), hipMemcpyDeviceToHost);

	for(y = 0; y < width*height; ++y) {

		out_data[y] = temp[y];

		if (temp[y] < 0)
		{
			out_data[y] = 0;
		}
		else if (temp[y] > 255)
		{
			out_data[y] = 255;
		}
	}
}

__global__ void dct_quantize_row(uint8_t *in_data, int16_t *temp, uint8_t *prediction, int w, int h, int16_t *out_data,
		int quantization)
{
 	int block_offset = blockIdx.x * 64;
 	int x_offset = blockIdx.x * 8;

 	int i = threadIdx.y;
 	int j = threadIdx.x;

	temp[block_offset + i * 8 + j] = ((int16_t) in_data[i * w + j + x_offset] - prediction[i * w + j + x_offset]);

 	dct_quant_block_8x8(temp, out_data, quantization, block_offset, i, j);
}


void dct_quantize(uint8_t *in_data, uint8_t *prediction, uint32_t width, uint32_t height,
		int16_t *out_data, int quantization)
{
	int numBlocks = width/8;
	dim3 threadsPerBlock(8, 8);

	unsigned int y;
	for (y = 0; y < height; y += 8)
	{
		dct_quantize_row<<<numBlocks, threadsPerBlock>>>(in_data + y * width, gpu_frame1_16 + y * width, prediction + y * width, width, height,
				gpu_out_16 + y * width, quantization);
	}
	hipMemcpy(out_data, gpu_out_16, width*height*sizeof(int16_t), hipMemcpyDeviceToHost);
}

void destroy_frame(struct frame *f)
{
	/* First frame doesn't have a reconstructed frame to destroy */
	if (!f)
	{
		return;
	}

	free(f->recons->Y);
	free(f->recons->U);
	free(f->recons->V);
	free(f->recons);

	free(f->residuals->Ydct);
	free(f->residuals->Udct);
	free(f->residuals->Vdct);
	free(f->residuals);

	free(f->predicted->Y);
	free(f->predicted->U);
	free(f->predicted->V);
	free(f->predicted);

	free(f->mbs[Y_COMPONENT]);
	free(f->mbs[U_COMPONENT]);
	free(f->mbs[V_COMPONENT]);

	free(f);
}

struct frame* create_frame(struct c63_common *cm, yuv_t *image)
{
	struct frame *f = (frame*) malloc(sizeof(struct frame));

	f->orig = image;

	f->recons = (yuv_t*) malloc(sizeof(yuv_t));
	f->recons->Y = (uint8_t*) malloc(cm->ypw * cm->yph);
	f->recons->U = (uint8_t*) malloc(cm->upw * cm->uph);
	f->recons->V = (uint8_t*) malloc(cm->vpw * cm->vph);

	f->predicted = (yuv_t*) malloc(sizeof(yuv_t));
	f->predicted->Y = (uint8_t*) calloc(cm->ypw * cm->yph, sizeof(uint8_t));
	f->predicted->U = (uint8_t*) calloc(cm->upw * cm->uph, sizeof(uint8_t));
	f->predicted->V = (uint8_t*) calloc(cm->vpw * cm->vph, sizeof(uint8_t));

	f->residuals = (dct_t*) malloc(sizeof(dct_t));
	f->residuals->Ydct = (int16_t*) calloc(cm->ypw * cm->yph, sizeof(int16_t));
	f->residuals->Udct = (int16_t*) calloc(cm->upw * cm->uph, sizeof(int16_t));
	f->residuals->Vdct = (int16_t*) calloc(cm->vpw * cm->vph, sizeof(int16_t));

	f->mbs[Y_COMPONENT] = (macroblock*) calloc(cm->mb_rows * cm->mb_cols,
			sizeof(struct macroblock));
	f->mbs[U_COMPONENT] = (macroblock*) calloc(cm->mb_rows / 2 * cm->mb_cols / 2,
			sizeof(struct macroblock));
	f->mbs[V_COMPONENT] = (macroblock*) calloc(cm->mb_rows / 2 * cm->mb_cols / 2,
			sizeof(struct macroblock));

	return f;
}

void dump_image(yuv_t *image, int w, int h, FILE *fp)
{
	fwrite(image->Y, 1, w * h, fp);
	fwrite(image->U, 1, w * h / 4, fp);
	fwrite(image->V, 1, w * h / 4, fp);
}
