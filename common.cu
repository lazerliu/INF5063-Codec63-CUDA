#include "hip/hip_runtime.h"
extern "C" {
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
}

#include "common.h"
#include "dsp.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int16_t *gpu_frame1_16;
int16_t *gpu_frame2_16;
int16_t *gpu_out_16;

uint8_t *gpu_out_8;
uint8_t *gpu_in_8;
uint8_t *gpu_prediction_8;

void cuda_init(int width, int height){
	hipMalloc(&gpu_frame1_16, width*height*sizeof(int16_t));
	hipMalloc(&gpu_frame2_16, width*height*sizeof(int16_t));
	hipMalloc(&gpu_out_16, width*height*sizeof(int16_t));

	hipMalloc(&gpu_out_8, width*height*sizeof(uint8_t));
	hipMalloc(&gpu_in_8, width*height*sizeof(uint8_t));
	hipMalloc(&gpu_prediction_8, width*height*sizeof(uint8_t));
}

void cuda_cleanup() {
	hipFree(gpu_frame1_16);
	hipFree(gpu_frame2_16);
	hipFree(gpu_out_16);

	hipFree(gpu_in_8);
	hipFree(gpu_out_8);
	hipFree(gpu_prediction_8);
}

void dequantize_end(int16_t *in_data, uint8_t *prediction, int w, uint8_t *out_data) {

	int x;
	for (x = 0; x < w; x += 8)
	{

		int i, j;
		for (i = 0; i < 8; ++i)
		{
			for (j = 0; j < 8; ++j)
			{
				/* Add prediction block. Note: DCT is not precise -
		 	 	Clamp to legal values */
				int16_t tmp = in_data[i * 8 + j + x * 8] + (int16_t) prediction[i * w + j + x];

				if (tmp < 0)
				{
					tmp = 0;
				}
				else if (tmp > 255)
				{
					tmp = 255;
				}

				out_data[i * w + j + x] = tmp;
			}
		}
	}
}

__host__ void dequantize_idct_row(int16_t *in_data, uint8_t *prediction, int w, int h, int y,
		uint8_t *out_data, int quantization)
{
	int x;
	for(x = 0; x < w; x += 8) {
		dequant_idct_block_8x8(in_data + (x * 8), gpu_out_16 + (x * 8), quantization);
	}


	int16_t block[w*8];
	hipMemcpy(block, gpu_out_16, w*8*sizeof(int16_t), hipMemcpyDeviceToHost);

	// Perform the dequantization and iDCT
	for (x = 0; x < w; x += 8)
	{
		int i, j;

		for (i = 0; i < 8; ++i)
		{
			for (j = 0; j < 8; ++j)
			{
				// Add prediction block. Note: DCT is not precise -
				// Clamp to legal values
				int16_t tmp = block[i * 8 + j + x * 8] + (int16_t) prediction[i * w + j + x];

				if (tmp < 0)
				{
					tmp = 0;
				}
				else if (tmp > 255)
				{
					tmp = 255;
				}

				out_data[i * w + j + x] = tmp;
			}
		}
	}
}

void dequantize_idct(int16_t *in_data, uint8_t *prediction, uint32_t width, uint32_t height,
		uint8_t *out_data, int quantization)
{
	unsigned int y;

	hipMemcpy(gpu_frame1_16, in_data, width*height*sizeof(int16_t), hipMemcpyHostToDevice);

	for (y = 0; y < height; y += 8)
	{
		dequantize_idct_row(gpu_frame1_16 + y * width, prediction + y * width, width, height, y,
				out_data + y * width, quantization);
	}
}

__host__ void dct_quantize_row(int16_t *in_data, uint8_t *prediction, int w, int h, int16_t *out_data,
		int quantization)
{
	//int16_t row[w*8];
	//int num_blocks_8x8 = w / 64;
	//int num_rest_blocks = w % 64 / 8;
	//printf("num_blocks: %d  - rest_blocks: %d \n", num_blocks_8x8, num_rest_blocks);
	/*
	// Perform the DCT and quantization
	int x;
	for (x = 0; x < w; x += 8)
	{
		int i, j;

		for (i = 0; i < 8; ++i)
		{
			for (j = 0; j < 8; ++j)
			{
				row[i * 8 + j + x * 8] = ((int16_t) in_data[i * w + j + x] - prediction[i * w + j + x]);
			}
		}
	}

	hipMemcpy(gpu_frame1, &row, w*8*sizeof(int16_t), hipMemcpyHostToDevice);
	*/
	/*int numBlocks = 1;
	dim3 threadsPerBlock(8,8);

	int x;
	for (x = 0; x < w; x += 8) {
		dct_quant_block_8x8<<<numBlocks, threadsPerBlock>>>(in_data + (x * 8), out_data + (x * 8), quantization);
	}
	*/

	int numBlocks = 16;
	int pixelsPerBlock = 64;
	int pixelsPerRow = w * 8;
	int restBlocks = (pixelsPerRow % (numBlocks * pixelsPerBlock))/pixelsPerBlock;

	dim3 threadsPerBlock(8, 8);

	int x;
	for (x = 0; x < pixelsPerRow; x += numBlocks*pixelsPerBlock) {
		dct_quant_block_8x8<<<numBlocks, threadsPerBlock>>>(in_data + x, out_data + x, quantization);
	}
	dct_quant_block_8x8<<<restBlocks, threadsPerBlock>>>(in_data + (pixelsPerRow - (restBlocks*pixelsPerBlock)), out_data + (pixelsPerRow - (restBlocks*pixelsPerBlock)), quantization);

}

 __global__ void setup_data(uint8_t *in_data, uint8_t *prediction, int16_t *out_data, int w, int h) {
		/* Perform the DCT and quantization */
	 	int i = threadIdx.y;
	 	int j = threadIdx.x;

		int x;
		for (x = 0; x < w; x += 8)
		{
			//int i, j;

			//for (i = 0; i < 8; ++i)
			//{
				//for (j = 0; j < 8; ++j)
				//{
					out_data[i * 8 + j + x * 8] = ((int16_t) in_data[i * w + j + x] - prediction[i * w + j + x]);
				//}
			//}
		}
}

void dct_quantize(uint8_t *in_data, uint8_t *prediction, uint32_t width, uint32_t height,
		int16_t *out_data, int quantization)
{
	unsigned int y;
	hipMemcpy(gpu_in_8, in_data, width*height*sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(gpu_prediction_8, prediction, width*height*sizeof(uint8_t), hipMemcpyHostToDevice);
	//hipMemcpy(gpu_frame1_16, &frame, gpu_predictionwidth*height*sizeof(int16_t), hipMemcpyHostToDevice);

	int numBlocks = 1;
	dim3 threadsPerBlock(8, 8);

	int16_t frame[width*height];
	//hipMemcpy(gpu_frame1_16, &frame, width*height*sizeof(int16_t), hipMemcpyHostToDevice);

	for (y = 0; y < height; y += 8)
	{
		setup_data<<<numBlocks, threadsPerBlock>>>(gpu_in_8 + y * width, gpu_prediction_8 + y * width, gpu_frame1_16 + y * width, width, height);
		dct_quantize_row(gpu_frame1_16 + y * width, prediction + y * width, width, height,
				gpu_out_16 + y * width, quantization);
	}
	hipMemcpy(out_data, gpu_out_16, width*height*sizeof(int16_t), hipMemcpyDeviceToHost);
}

void destroy_frame(struct frame *f)
{
	/* First frame doesn't have a reconstructed frame to destroy */
	if (!f)
	{
		return;
	}

	free(f->recons->Y);
	free(f->recons->U);
	free(f->recons->V);
	free(f->recons);

	free(f->residuals->Ydct);
	free(f->residuals->Udct);
	free(f->residuals->Vdct);
	free(f->residuals);

	free(f->predicted->Y);
	free(f->predicted->U);
	free(f->predicted->V);
	free(f->predicted);

	free(f->mbs[Y_COMPONENT]);
	free(f->mbs[U_COMPONENT]);
	free(f->mbs[V_COMPONENT]);

	free(f);
}

struct frame* create_frame(struct c63_common *cm, yuv_t *image)
{
	struct frame *f = (frame*) malloc(sizeof(struct frame));

	f->orig = image;

	f->recons = (yuv_t*) malloc(sizeof(yuv_t));
	f->recons->Y = (uint8_t*) malloc(cm->ypw * cm->yph);
	f->recons->U = (uint8_t*) malloc(cm->upw * cm->uph);
	f->recons->V = (uint8_t*) malloc(cm->vpw * cm->vph);

	f->predicted = (yuv_t*) malloc(sizeof(yuv_t));
	f->predicted->Y = (uint8_t*) calloc(cm->ypw * cm->yph, sizeof(uint8_t));
	f->predicted->U = (uint8_t*) calloc(cm->upw * cm->uph, sizeof(uint8_t));
	f->predicted->V = (uint8_t*) calloc(cm->vpw * cm->vph, sizeof(uint8_t));

	f->residuals = (dct_t*) malloc(sizeof(dct_t));
	f->residuals->Ydct = (int16_t*) calloc(cm->ypw * cm->yph, sizeof(int16_t));
	f->residuals->Udct = (int16_t*) calloc(cm->upw * cm->uph, sizeof(int16_t));
	f->residuals->Vdct = (int16_t*) calloc(cm->vpw * cm->vph, sizeof(int16_t));

	f->mbs[Y_COMPONENT] = (macroblock*) calloc(cm->mb_rows * cm->mb_cols,
			sizeof(struct macroblock));
	f->mbs[U_COMPONENT] = (macroblock*) calloc(cm->mb_rows / 2 * cm->mb_cols / 2,
			sizeof(struct macroblock));
	f->mbs[V_COMPONENT] = (macroblock*) calloc(cm->mb_rows / 2 * cm->mb_cols / 2,
			sizeof(struct macroblock));

	return f;
}

void dump_image(yuv_t *image, int w, int h, FILE *fp)
{
	fwrite(image->Y, 1, w * h, fp);
	fwrite(image->U, 1, w * h / 4, fp);
	fwrite(image->V, 1, w * h / 4, fp);
}
