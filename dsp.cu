#include "hip/hip_runtime.h"
#include <inttypes.h>
#include <math.h>
#include <stdlib.h>

extern "C" {
#include "dsp.h"
#include "tables.h"
}


__constant__ float dct_lookup[64] = 
{
	1.00000000000000000000f, 0.98078525066375732422f, 0.92387950420379638672f, 0.83146959543228149414f, 0.70710676908493041992f, 0.55557024478912353516f, 0.38268342614173889160f, 0.19509032368659973145f,
	1.00000000000000000000f, 0.83146959543228149414f, 0.38268342614173889160f,-0.19509032368659973145f,-0.70710676908493041992f,-0.98078525066375732422f,-0.92387950420379638672f,-0.55557024478912353516f,
	1.00000000000000000000f, 0.55557024478912353516f,-0.38268342614173889160f,-0.98078525066375732422f,-0.70710676908493041992f, 0.19509032368659973145f, 0.92387950420379638672f, 0.83146959543228149414f,
	1.00000000000000000000f, 0.19509032368659973145f,-0.92387950420379638672f,-0.55557024478912353516f, 0.70710676908493041992f, 0.83146959543228149414f,-0.38268342614173889160f,-0.98078525066375732422f,
	1.00000000000000000000f,-0.19509032368659973145f,-0.92387950420379638672f, 0.55557024478912353516f, 0.70710676908493041992f,-0.83146959543228149414f,-0.38268342614173889160f, 0.98078525066375732422f,
	1.00000000000000000000f,-0.55557024478912353516f,-0.38268342614173889160f, 0.98078525066375732422f,-0.70710676908493041992f,-0.19509032368659973145f, 0.92387950420379638672f,-0.83146959543228149414f,
	1.00000000000000000000f,-0.83146959543228149414f, 0.38268342614173889160f, 0.19509032368659973145f,-0.70710676908493041992f, 0.98078525066375732422f,-0.92387950420379638672f, 0.55557024478912353516f,
	1.00000000000000000000f,-0.98078525066375732422f, 0.92387950420379638672f,-0.83146959543228149414f, 0.70710676908493041992f,-0.55557024478912353516f, 0.38268342614173889160f,-0.19509032368659973145f
};


/* Array containing the indexes resulting from calculating
 * (zigzag_V[zigzag]*8) + zigzag_U[zigzag] for zigzag = 0, 1, ..., 63
 */
__constant__ uint8_t UV_indexes[64] =
{
	 0,  1,  8, 16,  9,  2,  3, 10,
	17, 24, 32, 25, 18, 11,  4,  5,
	12, 19, 26, 33, 40, 48, 41, 34,
	27, 20, 13,  6,  7, 14, 21, 28,
	35, 42, 49, 56, 57, 50, 43, 36,
	29, 22, 15, 23, 30, 37, 44, 51,
	58, 59, 52, 45, 38, 31, 39, 46,
	53, 60, 61, 54, 47, 55, 62, 63,
};

__shared__ float dct_macro_block[64];
__shared__ float dct_macro_block2[64];
__shared__ float idct_macro_block[64];
__shared__ float idct_macro_block2[64];



float *cuda_mb, *cuda_mb2;
int16_t *cuda_in_data, *cuda_out_data;
uint8_t *cuda_quant_tbl;

__host__ void cuda_init() {
	
	hipMalloc(&cuda_mb, 64*sizeof(float));
	hipMalloc(&cuda_mb2, 64*sizeof(float));
	
	hipMalloc(&cuda_in_data, 64*sizeof(int16_t));
	hipMalloc(&cuda_out_data, 64*sizeof(int16_t));
	
	hipMalloc(&cuda_quant_tbl, 64*sizeof(uint8_t));
}

__host__ void cuda_cleanup() {
	hipFree(cuda_mb);
	hipFree(cuda_mb2);
	
	hipFree(cuda_in_data);
	hipFree(cuda_out_data);
	
	hipFree(cuda_quant_tbl);
}


__global__ void gpu_transpose_block() 
{
	
    int i = threadIdx.y;
    int j = threadIdx.x;
    dct_macro_block[i * 8 + j] = dct_macro_block2[j * 8 + i];
    /*
    int j;
    for(j = 0; j < 8; j++) {	
		out_data[i * 8 + j] = in_data[j * 8 + i];
	}*/
}
/*
__host__ void transpose_block(float *in_data, float *out_data)
{
	gpu_transpose_block<<<1, 8>>>(gpu_in, gpu_out);
	
	
	
	/*
	 * 
	 * float* d_A; 
	 * hipMalloc(&d_A, size);
	 * 
	 * float* d_B; 
	 * hipMalloc(&d_B, size); 
	 * 
	 * float* d_C; 
	 * hipMalloc(&d_C, size); 
	 * 
	 * // Copy vectors from host memory to device memory 
	 * hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice); 
	 * hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice); 
	 * 
	 * // Invoke kernel 
	 * int threadsPerBlock = 256; 
	 * int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock; 
	 * VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N); 
	 * 
	 * // Copy result from device memory to host memory 
	 * // h_C contains the result in host memory 
	 * hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost); 
	 * 
	 * // Free device memory 
	 * hipFree(d_A); 
	 * hipFree(d_B); 
	 * hipFree(d_C);
*/
	
	/*
	int i, j;
	

	for (i = 0; i < 8; ++i)
	{
		for (j = 0; j < 8; ++j)
		{
			out_data[i * 8 + j] = in_data[j * 8 + i];
		}
	}
	* 
	
}*/

/*
static void dct_1d(float *in_data, float *out_data)
{
	int i, j;

	for (i = 0; i < 8; ++i)
	{
		float dct = 0;

		for (j = 0; j < 8; ++j)
		{
			dct += in_data[j] * dctlookup[j*8+i];
		}

		out_data[i] = dct;
	}
}
* */

__global__ void gpu_dct_1d()
{
	int i = threadIdx.y;
	int j = threadIdx.x;
	
	float idct = 0;
	
	int k;
	for (k = 0; k < 8; ++k) {
		idct += dct_macro_block[i*8+k] * dct_lookup[k*8+j];
	}
	dct_macro_block2[i*8+j] = idct;
}

/*
static void idct_1d(float *in_data, float *out_data)
{
	int i, j;

	for (i = 0; i < 8; ++i)
	{
		float idct = 0;

		for (j = 0; j < 8; ++j)
		{
			idct += in_data[j] * dctlookup[i*8+j];
		}

		out_data[i] = idct;
	}
}
* */


__global__ void gpu_idct_1d()
{
	int i = threadIdx.y;
	int j = threadIdx.x;
	float idct = 0;
	
	int k;
	for (k = 0; k < 8; ++k) {
		idct += idct_macro_block2[i*8+k] * dct_lookup[j*8+k];
	}
	idct_macro_block[i*8+j] = idct;
}


static void scale_block(float *in_data, float *out_data)
{
	int u, v;

	for (v = 0; v < 8; ++v)
	{
		for (u = 0; u < 8; ++u)
		{
			float a1 = !u ? ISQRT2 : 1.0f;
			float a2 = !v ? ISQRT2 : 1.0f;

			/* Scale according to normalizing function */
			out_data[v * 8 + u] = in_data[v * 8 + u] * a1 * a2;
		}
	}
}


static void quantize_block(float *in_data, float *out_data, uint8_t *quant_tbl)
{
	int zigzag;

	for (zigzag = 0; zigzag < 64; ++zigzag)
	{
		uint8_t u = zigzag_U[zigzag];
		uint8_t v = zigzag_V[zigzag];

		float dct = in_data[v * 8 + u];

		/* Zig-zag and quantize */
		out_data[zigzag] = (float) round((dct / 4.0) / quant_tbl[zigzag]);
	}
}

__global__ void gpu_dequantize_block(float *in_data, float *out_data, uint8_t *quant_tbl)
{
	
	
}

static void dequantize_block(float *in_data, float *out_data, uint8_t *quant_tbl)
{
	int zigzag;

	for (zigzag = 0; zigzag < 64; ++zigzag)
	{
		uint8_t u = zigzag_U[zigzag];
		uint8_t v = zigzag_V[zigzag];

		float dct = in_data[zigzag];

		/* Zig-zag and de-quantize */
		out_data[v * 8 + u] = (float) round((dct * quant_tbl[zigzag]) / 4.0);
	}
}


__global__ void gpu_dct_quant_block_8x8(int16_t *in_data, int16_t *out_data, uint8_t *quant_tbl)
{
	int i = threadIdx.y;
	int j = threadIdx.x;
	
	
	// Copy pixel to shared memory
	dct_macro_block2[i*8+j] = in_data[i*8+j];
	__syncthreads();
	
	
	// First dct_1d - mb = mb2
	float dct = 0;
	int k;	
	for (k = 0; k < 8; ++k) {
		dct += dct_macro_block2[i*8+k] * dct_lookup[k*8+j];
	}
	dct_macro_block[i*8+j] = dct;
	__syncthreads();
	
	
	// First transpose - mb2 = mb
	dct_macro_block2[i * 8 + j] = dct_macro_block[j * 8 + i];
	__syncthreads();
	
	
	// Second dct_1d - mb = mb2
	dct = 0;
	for (k = 0; k < 8; ++k) {
		dct += dct_macro_block2[i*8+k] * dct_lookup[k*8+j];
	}
	dct_macro_block[i*8+j] = dct;
	__syncthreads();
	
	
	// Second transpose - mb2 = mb
	dct_macro_block2[i * 8 + j] = dct_macro_block[j * 8 + i];
	__syncthreads();
	
	
	// Scale
	dct_macro_block[i*8+j] = dct_macro_block2[i*8+j];
	if(i == 0) {
		dct_macro_block[i*8+j] *= ISQRT2;
	}
	
	if(j == 0) {
		dct_macro_block[i*8+j] *= ISQRT2;
	}
	__syncthreads();
	
	
	// Quantize	
	dct = dct_macro_block[UV_indexes[i*8+j]];
	dct_macro_block2[i*8+j] = (float) round((dct/4.0/quant_tbl[i*8+j]));
	__syncthreads();
	
	// Set value in cuda_out_data
	out_data[i*8+j] = dct_macro_block2[i*8+j];
	__syncthreads();
	
	/*// Copy to mb - temporary
	mb2[i*8+j] = dct_macro_block2[i*8+j];
	mb[i*8+j] = dct_macro_block[i*8+j];
	*/	
}

__host__ void dct_quant_block_8x8(int16_t *in_data, int16_t *out_data, uint8_t *quant_tbl)
{
	hipMemcpy(cuda_in_data, in_data , 64*sizeof(int16_t), hipMemcpyHostToDevice); 
	hipMemcpy(cuda_quant_tbl, quant_tbl, 64*sizeof(uint8_t), hipMemcpyHostToDevice);
	/*
	for (i = 0; i < 64; ++i)
	{
		mb2[i] = in_data[i];
	}*/
	
	int numBlocks = 1;
	dim3 threadsPerBlock(8, 8);
	
	gpu_dct_quant_block_8x8<<<numBlocks, threadsPerBlock>>>(cuda_in_data, cuda_out_data, cuda_quant_tbl);
	hipMemcpy(out_data, cuda_out_data, 64*sizeof(int16_t), hipMemcpyDeviceToHost);
	
	
	//gpu_copy_int_to_shared<<<numBlocks, threadsPerBlock>>>(in_data, out_data);
	
	
	//hipMemcpy(gpu_in, (float*)&mb2, 64*sizeof(float), hipMemcpyHostToDevice); 

	/* Two 1D DCT operations with transpose */
	/*
	for (v = 0; v < 8; ++v)
	{
		//dct_1d(mb2 + v * 8, mb + v * 8);
	}
	* */
	//gpu_dct_1d<<<numBlocks, threadsPerBlock>>>();

	//transpose_block(mb, mb2);
	//gpu_transpose_block<<<numBlocks, threadsPerBlock>>>();
	
	/*
	for (v = 0; v < 8; ++v)
	{
		//dct_1d(mb2 + v * 8, mb + v * 8);
	}
	* */
	//gpu_dct_1d<<<numBlocks, threadsPerBlock>>>();
	
	//gpu_transpose_block<<<numBlocks, threadsPerBlock>>>();
	//transpose_block(mb, mb2);
	
	//gpu_copy_to_mb<<<numBlocks, threadsPerBlock>>>(mb2, mb);
	
	/*float mb[8*8];
	float mb2[8*8];
	
	hipMemcpy((float*)&mb2, cuda_mb2, 64*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy((float*)&mb, cuda_mb, 64*sizeof(float), hipMemcpyDeviceToHost);

	int i;
	for (i = 0; i < 64; ++i)
	{
		out_data[i] = mb2[i];
	}*/
}


__global__ void gpu_dequant_idct_block_8x8(int16_t *in_data, int16_t *out_data, uint8_t *quant_tbl)
{
	int i = threadIdx.y;
	int j = threadIdx.x;
	
	// Copy to shared memory
	idct_macro_block[i*8+j] = in_data[i*8+j];
	__syncthreads();
	
	
	// Dequantize
	float dct = idct_macro_block[i*8+j];
	idct_macro_block2[UV_indexes[i*8+j]] = (float) round((dct*quant_tbl[i*8+j]) / 4.0);
	__syncthreads();
	
	
	// Scale
	idct_macro_block[i*8+j] = idct_macro_block2[i*8+j];
	if(i == 0) {
		idct_macro_block[i*8+j] *= ISQRT2;
	}
	if(j == 0) {
		idct_macro_block[i*8+j] *= ISQRT2;
	}
	__syncthreads();
		
	
	// First idct - mb2 = mb
	float idct = 0;
	int k;
	for (k = 0; k < 8; ++k) {
		idct += idct_macro_block[i*8+k] * dct_lookup[j*8+k];
	}
	idct_macro_block2[i*8+j] = idct;
	__syncthreads();
	
	
	// First transpose - mb = mb2
	idct_macro_block[i * 8 + j] = idct_macro_block2[j * 8 + i];
	__syncthreads();
	
	// Second idct - mb2 = mb
	idct = 0;
	for (k = 0; k < 8; ++k) {
		idct += idct_macro_block[i*8+k] * dct_lookup[j*8+k];
	}
	idct_macro_block2[i*8+j] = idct;
	__syncthreads();
	
	
	// Second transpose - mb = mb2
	idct_macro_block[i * 8 + j] = idct_macro_block2[j * 8 + i];
	__syncthreads();
	
	
	// Copy to out_data
	out_data[i*8+j] = idct_macro_block[i*8+j];
	__syncthreads();
}

__host__ void dequant_idct_block_8x8(int16_t *in_data, int16_t *out_data, uint8_t *quant_tbl)
{
	/*float mb[8 * 8];
	float mb2[8 * 8];

	int i;

	for (i = 0; i < 64; ++i)
	{
		mb[i] = in_data[i];
	}*/

	//dequantize_block(mb, mb2, quant_tbl);

	hipMemcpy(cuda_in_data, in_data, 64*sizeof(int16_t), hipMemcpyHostToDevice);
	hipMemcpy(cuda_quant_tbl, quant_tbl, 64*sizeof(uint8_t), hipMemcpyHostToDevice);
	
	int numBlocks = 1;
	dim3 threadsPerBlock(8, 8);
	
	gpu_dequant_idct_block_8x8<<<numBlocks, threadsPerBlock>>>(cuda_in_data, cuda_out_data, cuda_quant_tbl);
	
	hipMemcpy(out_data, cuda_out_data, 64*sizeof(int16_t), hipMemcpyDeviceToHost);
	
	
	//gpu_copy_float_to_shared<<<numBlocks, threadsPerBlock>>>(mb, mb2);
	
	/*hipMemcpy(gpu_in, (float*)&mb, 64*sizeof(float), hipMemcpyHostToDevice); 
	hipMemcpy(gpu_out, (float*)&mb2, 64*sizeof(float), hipMemcpyHostToDevice);
	*/
	
	/* Two 1D inverse DCT operations with transpose */
	/*
	for (v = 0; v < 8; ++v)
	{
		//idct_1d(mb + v * 8, mb2 + v * 8);
		gpu_idct_1d<<<numBlocks, 8>>>(gpu_in + v * 8, gpu_out + v * 8);
	}
	* */
	//gpu_idct_1d<<<numBlocks, threadsPerBlock>>>();
	
	//transpose_block(mb2, mb);
	//gpu_transpose_block<<<numBlocks, threadsPerBlock>>>();
	
	/*
	for (v = 0; v < 8; ++v)
	{
		//idct_1d(mb + v * 8, mb2 + v * 8);
		gpu_idct_1d<<<numBlocks, 8>>>(gpu_in + v * 8, gpu_out + v * 8);
	}
	* */	
	//gpu_idct_1d<<<numBlocks, threadsPerBlock>>>();
	
	//transpose_block(mb2, mb);
	//gpu_transpose_block<<<numBlocks, threadsPerBlock>>>();
	
	//gpu_copy_to_int<<<numBlocks, threadsPerBlock>>>(mb);

	
	/*hipMemcpy((float*)&mb, gpu_in, 64*sizeof(float), hipMemcpyDeviceToHost);
	
	for (i = 0; i < 64; ++i)
	{
		out_data[i] = mb[i];
	}*/
}

void sad_block_8x8(uint8_t *block1, uint8_t *block2, int stride, int *result)
{
	int u, v;

	*result = 0;

	for (v = 0; v < 8; ++v)
	{
		for (u = 0; u < 8; ++u)
		{
			*result += abs(block2[v * stride + u] - block1[v * stride + u]);
		}
	}
}
