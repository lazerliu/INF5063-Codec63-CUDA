#include "hip/hip_runtime.h"
extern "C" {
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "c63.h"
#include "c63_write.h"
#include "tables.h"
}

#include "common.h"
#include "me.h"


static char *output_file, *input_file;
FILE *outfile;

static int limit_numframes = 0;

static uint32_t width;
static uint32_t height;

/* getopt */
extern int optind;
extern char *optarg;

// Get CPU cycle count
uint64_t rdtsc(){
    unsigned int lo,hi;
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return ((uint64_t)hi << 32) | lo;
}

/* Read planar YUV frames with 4:2:0 chroma sub-sampling */
static yuv_t* read_yuv(FILE *file, struct c63_common *cm)
{
  size_t len = 0;
  yuv_t *image = (yuv_t*) malloc(sizeof(*image));

  /* Read Y. The size of Y is the same as the size of the image. The indices
     represents the color component (0 is Y, 1 is U, and 2 is V) */
  image->Y = (uint8_t*) calloc(1, cm->padw[Y_COMPONENT]*cm->padh[Y_COMPONENT]);
  len += fread(image->Y, 1, width*height, file);

  /* Read U. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y
     because (height/2)*(width/2) = (height*width)/4. */
  image->U = (uint8_t*) calloc(1, cm->padw[U_COMPONENT]*cm->padh[U_COMPONENT]);
  len += fread(image->U, 1, (width*height)/4, file);

  /* Read V. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y. */
  image->V = (uint8_t*) calloc(1, cm->padw[V_COMPONENT]*cm->padh[V_COMPONENT]);
  len += fread(image->V, 1, (width*height)/4, file);

  if (ferror(file))
  {
    perror("ferror");
    exit(EXIT_FAILURE);
  }

  if (feof(file))
  {
    free(image->Y);
    free(image->U);
    free(image->V);
    free(image);

    return NULL;
  }
  else if (len != width*height*1.5)
  {
    fprintf(stderr, "Reached end of file, but incorrect bytes read.\n");
    fprintf(stderr, "Wrong input? (height: %d width: %d)\n", height, width);

    free(image->Y);
    free(image->U);
    free(image->V);
    free(image);

    return NULL;
  }

  return image;
}

static void c63_encode_image(struct c63_common *cm, yuv_t *image)
{
  /* Advance to next frame */
  destroy_frame(cm->refframe);
  cm->refframe = cm->curframe;
  cm->curframe = create_frame(cm, image);

  /* Check if keyframe */
  if (cm->framenum == 0 || cm->frames_since_keyframe == cm->keyframe_interval)
  {
    cm->curframe->keyframe = 1;
    cm->frames_since_keyframe = 0;

    fprintf(stderr, " (keyframe) ");
  }
  else { cm->curframe->keyframe = 0; }

  if (!cm->curframe->keyframe)
  {
    /* Motion Estimation */
    c63_motion_estimate(cm);

    /* Motion Compensation */
    c63_motion_compensate(cm);
  }

  /* DCT and Quantization */
  dct_quantize(image->Y, cm->curframe->predicted->Y, cm->padw[Y_COMPONENT],
      cm->padh[Y_COMPONENT], cm->curframe->residuals->Ydct,
      cm->quanttbl[Y_COMPONENT]);

  dct_quantize(image->U, cm->curframe->predicted->U, cm->padw[U_COMPONENT],
      cm->padh[U_COMPONENT], cm->curframe->residuals->Udct,
      cm->quanttbl[U_COMPONENT]);

  dct_quantize(image->V, cm->curframe->predicted->V, cm->padw[V_COMPONENT],
      cm->padh[V_COMPONENT], cm->curframe->residuals->Vdct,
      cm->quanttbl[V_COMPONENT]);

  /* Reconstruct frame for inter-prediction */
  dequantize_idct(cm->curframe->residuals->Ydct, cm->curframe->predicted->Y,
      cm->ypw, cm->yph, cm->curframe->recons->Y, cm->quanttbl[Y_COMPONENT]);
  dequantize_idct(cm->curframe->residuals->Udct, cm->curframe->predicted->U,
      cm->upw, cm->uph, cm->curframe->recons->U, cm->quanttbl[U_COMPONENT]);
  dequantize_idct(cm->curframe->residuals->Vdct, cm->curframe->predicted->V,
      cm->vpw, cm->vph, cm->curframe->recons->V, cm->quanttbl[V_COMPONENT]);

  /* Function dump_image(), found in common.c, can be used here to check if the
     prediction is correct */

  write_frame(cm);

  ++cm->framenum;
  ++cm->frames_since_keyframe;
}

static void set_cuda_searchrange_boundaries(c63_common* cm)
{
	int hY = cm->padh[Y_COMPONENT];
	int hUV = cm->padh[U_COMPONENT];

	int wY = cm->padw[Y_COMPONENT];
	int wUV = cm->padw[U_COMPONENT];

	int* leftsY = new int[cm->mb_cols];
	int* leftsUV = new int[cm->mb_cols/2];
	int* rightsY = new int[cm->mb_cols];
	int* rightsUV = new int[cm->mb_cols/2];
	int* topsY = new int[cm->mb_rows];
	int* topsUV = new int[cm->mb_rows/2];
	int* bottomsY = new int[cm->mb_rows];
	int* bottomsUV = new int[cm->mb_rows/2];

	for (int mb_x = 0; mb_x < cm->mb_cols; ++mb_x) {
		leftsY[mb_x] = mb_x*8 - 16;
		rightsY[mb_x] = mb_x*8 + 16;

		if (leftsY[mb_x] < 0) {
			leftsY[mb_x] = 0;
		}

		if (rightsY[mb_x] > (wY - 8)) {
			rightsY[mb_x] = wY - 8;
		}
	}

	for (int mb_x = 0; mb_x < cm->mb_cols/2; ++mb_x) {
		leftsUV[mb_x] = mb_x*8 - 8;
		rightsUV[mb_x] = mb_x*8 + 8;

		if (leftsUV[mb_x] < 0) {
			leftsUV[mb_x] = 0;
		}

		if (rightsUV[mb_x] > (wUV - 8)) {
			rightsUV[mb_x] = wUV - 8;
		}
	}

	for (int mb_y = 0; mb_y < cm->mb_rows; ++mb_y) {
		topsY[mb_y] = mb_y * 8 - 16;
		bottomsY[mb_y] = mb_y * 8 + 16;

		if (topsY[mb_y] < 0) {
			topsY[mb_y] = 0;
		}

		if (bottomsY[mb_y] > (hY - 8)) {
			bottomsY[mb_y] = hY - 8;
		}
	}

	for (int mb_y = 0; mb_y < cm->mb_rows/2; ++mb_y) {
		topsUV[mb_y] = mb_y * 8 - 8;
		bottomsUV[mb_y] = mb_y * 8 + 8;

		if (topsUV[mb_y] < 0) {
			topsUV[mb_y] = 0;
		}

		if (bottomsUV[mb_y] > (hUV - 8)) {
			bottomsUV[mb_y] = hUV - 8;
		}
	}

	hipMemcpy(cm->cuda_me.leftsY_gpu, leftsY, cm->mb_cols * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.leftsUV_gpu, leftsUV, cm->mb_cols/2 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.rightsY_gpu, rightsY, cm->mb_cols * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.rightsUV_gpu, rightsUV, cm->mb_cols/2 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.topsY_gpu, topsY, cm->mb_rows * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.topsUV_gpu, topsUV, cm->mb_rows/2 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.bottomsY_gpu, bottomsY, cm->mb_rows * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.bottomsUV_gpu, bottomsUV, cm->mb_rows/2 * sizeof(int), hipMemcpyHostToDevice);

	delete[] leftsY;
	delete[] leftsUV;
	delete[] rightsY;
	delete[] rightsUV;
	delete[] topsY;
	delete[] topsUV;
	delete[] bottomsY;
	delete[] bottomsUV;
}

static void init_cuda_data(c63_common* cm)
{
	cuda_data_me* cuda_me = &(cm->cuda_me);

	const int frame_size_Y = cm->padw[Y_COMPONENT] * cm->padh[Y_COMPONENT] * sizeof(uint8_t);
	const int frame_size_U = cm->padw[U_COMPONENT] * cm->padh[U_COMPONENT] * sizeof(uint8_t);
	const int frame_size_V = cm->padw[V_COMPONENT] * cm->padh[V_COMPONENT] * sizeof(uint8_t);

	hipMalloc((void**) &(cuda_me->origY_gpu), frame_size_Y);
	hipMalloc((void**) &(cuda_me->origU_gpu), frame_size_U);
	hipMalloc((void**) &(cuda_me->origV_gpu), frame_size_V);

	hipMalloc((void**) &(cuda_me->refY_gpu), frame_size_Y);
	hipMalloc((void**) &(cuda_me->refU_gpu), frame_size_U);
	hipMalloc((void**) &(cuda_me->refV_gpu), frame_size_V);

	const int vector_size = cm->mb_rows*cm->mb_cols*sizeof(int);

	cuda_me->vector_x = new int[cm->mb_rows * cm->mb_cols];
	cuda_me->vector_y = new int[cm->mb_rows * cm->mb_cols];

	hipMalloc((void**) &(cuda_me->vector_x_gpu), vector_size);
	hipMalloc((void**) &(cuda_me->vector_y_gpu), vector_size);

	hipMalloc((void**) &(cuda_me->leftsY_gpu), cm->mb_cols * sizeof(int));
	hipMalloc((void**) &(cuda_me->leftsUV_gpu), (cm->mb_cols/2) * sizeof(int));
	hipMalloc((void**) &(cuda_me->rightsY_gpu), cm->mb_cols * sizeof(int));
	hipMalloc((void**) &(cuda_me->rightsUV_gpu), (cm->mb_cols/2) * sizeof(int));
	hipMalloc((void**) &(cuda_me->topsY_gpu), cm->mb_rows * sizeof(int));
	hipMalloc((void**) &(cuda_me->topsUV_gpu), (cm->mb_rows/2) * sizeof(int));
	hipMalloc((void**) &(cuda_me->bottomsY_gpu), cm->mb_rows * sizeof(int));
	hipMalloc((void**) &(cuda_me->bottomsUV_gpu), (cm->mb_rows/2) * sizeof(int));

	set_cuda_searchrange_boundaries(cm);
}

static void cleanup_cuda_data(c63_common* cm)
{
	hipFree(cm->cuda_me.origY_gpu);
	hipFree(cm->cuda_me.origU_gpu);
	hipFree(cm->cuda_me.origV_gpu);

	hipFree(cm->cuda_me.refY_gpu);
	hipFree(cm->cuda_me.refU_gpu);
	hipFree(cm->cuda_me.refV_gpu);

	delete[] cm->cuda_me.vector_x;
	delete[] cm->cuda_me.vector_y;

	hipFree(cm->cuda_me.vector_x_gpu);
	hipFree(cm->cuda_me.vector_y_gpu);

	hipFree(cm->cuda_me.leftsY_gpu);
	hipFree(cm->cuda_me.leftsUV_gpu);
	hipFree(cm->cuda_me.rightsY_gpu);
	hipFree(cm->cuda_me.rightsUV_gpu);
	hipFree(cm->cuda_me.topsY_gpu);
	hipFree(cm->cuda_me.topsUV_gpu);
	hipFree(cm->cuda_me.bottomsY_gpu);
	hipFree(cm->cuda_me.bottomsUV_gpu);
}

struct c63_common* init_c63_enc(int width, int height)
{
  int i;

  /* calloc() sets allocated memory to zero */
  struct c63_common *cm = (c63_common*) calloc(1, sizeof(struct c63_common));

  cm->width = width;
  cm->height = height;

  cm->padw[Y_COMPONENT] = cm->ypw = (uint32_t)(ceil(width/16.0f)*16);
  cm->padh[Y_COMPONENT] = cm->yph = (uint32_t)(ceil(height/16.0f)*16);
  cm->padw[U_COMPONENT] = cm->upw = (uint32_t)(ceil(width*UX/(YX*8.0f))*8);
  cm->padh[U_COMPONENT] = cm->uph = (uint32_t)(ceil(height*UY/(YY*8.0f))*8);
  cm->padw[V_COMPONENT] = cm->vpw = (uint32_t)(ceil(width*VX/(YX*8.0f))*8);
  cm->padh[V_COMPONENT] = cm->vph = (uint32_t)(ceil(height*VY/(YY*8.0f))*8);

  cm->mb_cols = cm->ypw / 8;
  cm->mb_rows = cm->yph / 8;

  /* Quality parameters -- Home exam deliveries should have original values,
   i.e., quantization factor should be 25, search range should be 16, and the
   keyframe interval should be 100. */
  cm->qp = 25;                  // Constant quantization factor. Range: [1..50]
  cm->me_search_range = 16;     // Pixels in every direction
  cm->keyframe_interval = 100;  // Distance between keyframes

  /* Initialize quantization tables */
  for (i = 0; i < 64; ++i)
  {
    cm->quanttbl[Y_COMPONENT][i] = yquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[U_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[V_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
  }

  init_cuda_data(cm);

  return cm;
}

static void print_help()
{
  printf("Usage: ./c63enc [options] input_file\n");
  printf("Commandline options:\n");
  printf("  -h                             Height of images to compress\n");
  printf("  -w                             Width of images to compress\n");
  printf("  -o                             Output file (.c63)\n");
  printf("  [-f]                           Limit number of frames to encode\n");
  printf("\n");

  exit(EXIT_FAILURE);
}

int main(int argc, char **argv)
{
  int c;
  yuv_t *image;

  if (argc == 1) { print_help(); }

  while ((c = getopt(argc, argv, "h:w:o:f:i:")) != -1)
  {
    switch (c)
    {
      case 'h':
        height = atoi(optarg);
        break;
      case 'w':
        width = atoi(optarg);
        break;
      case 'o':
        output_file = optarg;
        break;
      case 'f':
        limit_numframes = atoi(optarg);
        break;
      default:
        print_help();
        break;
    }
  }

  if (optind >= argc)
  {
    fprintf(stderr, "Error getting program options, try --help.\n");
    exit(EXIT_FAILURE);
  }

  outfile = fopen(output_file, "wb");

  if (outfile == NULL)
  {
    perror("fopen");
    exit(EXIT_FAILURE);
  }

  struct c63_common *cm = init_c63_enc(width, height);
  cm->e_ctx.fp = outfile;

  input_file = argv[optind];

  if (limit_numframes) { printf("Limited to %d frames.\n", limit_numframes); }

  FILE *infile = fopen(input_file, "rb");

  if (infile == NULL)
  {
    perror("fopen");
    exit(EXIT_FAILURE);
  }

  /* Encode input frames */
  int numframes = 0;

# ifdef SHOW_CYCLES
  uint64_t kCycleCountTotal = 0;
# endif

  while (1)
  {
    image = read_yuv(infile, cm);

    if (!image) { break; }

    printf("Encoding frame %d, ", numframes);

# ifdef SHOW_CYCLES
    uint64_t cycleCountBefore = rdtsc();
    c63_encode_image(cm, image);
    uint64_t cycleCountAfter = rdtsc();

    uint64_t kCycleCount = (cycleCountAfter - cycleCountBefore)/1000;
    kCycleCountTotal += kCycleCount;
    printf("%" PRIu64 "k cycles, ", kCycleCount);
# else
    c63_encode_image(cm, image);
# endif

    free(image->Y);
    free(image->U);
    free(image->V);
    free(image);

    printf("Done!\n");

    ++numframes;

    if (limit_numframes && numframes >= limit_numframes) { break; }
  }

# ifdef SHOW_CYCLES
  printf("-----------\n");
  printf("Average CPU cycle count per frame: %" PRIu64 "k\n", kCycleCountTotal/numframes);
# endif

  cleanup_cuda_data(cm);

  fclose(outfile);
  fclose(infile);

  return EXIT_SUCCESS;
}
