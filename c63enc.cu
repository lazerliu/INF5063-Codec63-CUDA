#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <utility>

extern "C" {
#include "tables.h"
}

#include "c63.h"
#include "c63_write.h"
#include "common.h"
#include "dsp.h"
#include "me.h"

static char *output_file, *input_file;
FILE *outfile;

static int limit_numframes = 0;

static uint32_t width;
static uint32_t height;

/* getopt */
extern int optind;
extern char *optarg;


// Get CPU cycle count
uint64_t rdtsc(){
    unsigned int lo,hi;
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return ((uint64_t)hi << 32) | lo;
}

/* Read planar YUV frames with 4:2:0 chroma sub-sampling */
static bool read_yuv(FILE *file, yuv_t* image)
{
  size_t len = 0;

  /* Read Y. The size of Y is the same as the size of the image. The indices
     represents the color component (0 is Y, 1 is U, and 2 is V) */
  len += fread(image->Y, 1, width*height, file);

  /* Read U. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y
     because (height/2)*(width/2) = (height*width)/4. */
  len += fread(image->U, 1, (width*height)/4, file);

  /* Read V. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y. */
  len += fread(image->V, 1, (width*height)/4, file);

  if (ferror(file))
  {
    perror("ferror");
    exit(EXIT_FAILURE);
  }

  if (feof(file))
  {
    return false;
  }
  else if (len != width*height*1.5)
  {
    fprintf(stderr, "Reached end of file, but incorrect bytes read.\n");
    fprintf(stderr, "Wrong input? (height: %d width: %d)\n", height, width);

    return false;
  }

  return true;
}

static void zero_out_prediction(struct c63_common* cm)
{
	struct frame* frame = cm->curframe;
	hipMemset(frame->predicted_gpu->Y, 0, cm->ypw * cm->yph * sizeof(uint8_t));
	hipMemset(frame->predicted_gpu->U, 0, cm->upw * cm->uph * sizeof(uint8_t));
	hipMemset(frame->predicted_gpu->V, 0, cm->vpw * cm->vph * sizeof(uint8_t));
}

static void c63_encode_image(struct c63_common *cm, yuv_t* image_gpu)
{
	// Advance to next frame by swapping current and reference frame
	std::swap(cm->curframe, cm->refframe);

	cm->curframe->orig_gpu = image_gpu;

	/* Check if keyframe */
	if (cm->framenum == 0 || cm->frames_since_keyframe == cm->keyframe_interval)
	{
		cm->curframe->keyframe = 1;
		cm->frames_since_keyframe = 0;

		fprintf(stderr, " (keyframe) ");
	}
	else { cm->curframe->keyframe = 0; }

	if (!cm->curframe->keyframe)
	{
		/* Motion Estimation */
		c63_motion_estimate(cm);

		/* Motion Compensation */
		c63_motion_compensate(cm);
	}
	else
	{
		// dct_quantize() expects zeroed out prediction buffers for key frames.
		// We zero them out here since we reuse the buffers from previous frames.
		zero_out_prediction(cm);
	}

	yuv_t* predicted = cm->curframe->predicted_gpu;
	dct_t* residuals = cm->curframe->residuals_gpu;

	const dim3 threadsPerBlock(8, 8);

	const dim3 numBlocks_Y(cm->padw[Y_COMPONENT]/threadsPerBlock.x, cm->padh[Y_COMPONENT]/threadsPerBlock.y);
	const dim3 numBlocks_U(cm->padw[U_COMPONENT]/threadsPerBlock.x, cm->padh[U_COMPONENT]/threadsPerBlock.y);
	const dim3 numBlocks_V(cm->padw[V_COMPONENT]/threadsPerBlock.x, cm->padh[V_COMPONENT]/threadsPerBlock.y);

	/* DCT and Quantization */
	dct_quantize<<<numBlocks_Y, threadsPerBlock, 0, cm->cuda_data.streamY>>>(cm->curframe->orig_gpu->Y, predicted->Y,
			cm->padw[Y_COMPONENT], residuals->Ydct, Y_COMPONENT);
	hipMemcpyAsync(cm->curframe->residuals->Ydct, residuals->Ydct, cm->padw[Y_COMPONENT]*cm->padh[Y_COMPONENT]*sizeof(int16_t),
			hipMemcpyDeviceToHost, cm->cuda_data.streamY);

	dct_quantize<<<numBlocks_U, threadsPerBlock, 0, cm->cuda_data.streamU>>>(cm->curframe->orig_gpu->U, predicted->U,
			cm->padw[U_COMPONENT], residuals->Udct, U_COMPONENT);
	hipMemcpyAsync(cm->curframe->residuals->Udct, residuals->Udct, cm->padw[U_COMPONENT]*cm->padh[U_COMPONENT]*sizeof(int16_t),
			hipMemcpyDeviceToHost, cm->cuda_data.streamU);

	dct_quantize<<<numBlocks_V, threadsPerBlock, 0, cm->cuda_data.streamV>>>(cm->curframe->orig_gpu->V, predicted->V,
			cm->padw[V_COMPONENT], residuals->Vdct, V_COMPONENT);
	hipMemcpyAsync(cm->curframe->residuals->Vdct, residuals->Vdct, cm->padw[V_COMPONENT]*cm->padh[V_COMPONENT]*sizeof(int16_t),
			hipMemcpyDeviceToHost, cm->cuda_data.streamV);

	/* Reconstruct frame for inter-prediction */
	dequantize_idct<<<numBlocks_Y, threadsPerBlock, 0, cm->cuda_data.streamY>>>(residuals->Ydct, predicted->Y,
			cm->ypw, cm->curframe->recons_gpu->Y, Y_COMPONENT);

	dequantize_idct<<<numBlocks_U, threadsPerBlock, 0, cm->cuda_data.streamU>>>(residuals->Udct, predicted->U,
			cm->upw, cm->curframe->recons_gpu->U, U_COMPONENT);

	dequantize_idct<<<numBlocks_V, threadsPerBlock, 0, cm->cuda_data.streamV>>>(residuals->Vdct, predicted->V,
			cm->vpw, cm->curframe->recons_gpu->V, V_COMPONENT);

	/* Function dump_image(), found in common.c, can be used here to check if the
     prediction is correct */
}

static void set_searchrange_boundaries_cuda(c63_common* cm)
{
	int hY = cm->padh[Y_COMPONENT];
	int hUV = cm->padh[U_COMPONENT];

	int wY = cm->padw[Y_COMPONENT];
	int wUV = cm->padw[U_COMPONENT];

	int* leftsY = new int[cm->mb_colsY];
	int* leftsUV = new int[cm->mb_colsUV];
	int* rightsY = new int[cm->mb_colsY];
	int* rightsUV = new int[cm->mb_colsUV];
	int* topsY = new int[cm->mb_rowsY];
	int* topsUV = new int[cm->mb_rowsUV];
	int* bottomsY = new int[cm->mb_rowsY];
	int* bottomsUV = new int[cm->mb_rowsUV];

	for (int mb_x = 0; mb_x < cm->mb_colsY; ++mb_x) {
		leftsY[mb_x] = mb_x * 8 - ME_RANGE_Y;
		rightsY[mb_x] = mb_x * 8 + ME_RANGE_Y;

		if (leftsY[mb_x] < 0) {
			leftsY[mb_x] = 0;
		}

		if (rightsY[mb_x] > (wY - 8)) {
			rightsY[mb_x] = wY - 8;
		}
	}

	for (int mb_x = 0; mb_x < cm->mb_colsUV; ++mb_x) {
		leftsUV[mb_x] = mb_x * 8 - ME_RANGE_UV;
		rightsUV[mb_x] = mb_x * 8 + ME_RANGE_UV;

		if (leftsUV[mb_x] < 0) {
			leftsUV[mb_x] = 0;
		}

		if (rightsUV[mb_x] > (wUV - 8)) {
			rightsUV[mb_x] = wUV - 8;
		}
	}

	for (int mb_y = 0; mb_y < cm->mb_rowsY; ++mb_y) {
		topsY[mb_y] = mb_y * 8 - ME_RANGE_Y;
		bottomsY[mb_y] = mb_y * 8 + ME_RANGE_Y;

		if (topsY[mb_y] < 0) {
			topsY[mb_y] = 0;
		}

		if (bottomsY[mb_y] > (hY - 8)) {
			bottomsY[mb_y] = hY - 8;
		}
	}

	for (int mb_y = 0; mb_y < cm->mb_rowsUV; ++mb_y) {
		topsUV[mb_y] = mb_y * 8 - ME_RANGE_UV;
		bottomsUV[mb_y] = mb_y * 8 + ME_RANGE_UV;

		if (topsUV[mb_y] < 0) {
			topsUV[mb_y] = 0;
		}

		if (bottomsUV[mb_y] > (hUV - 8)) {
			bottomsUV[mb_y] = hUV - 8;
		}
	}

	hipMemcpy(cm->cuda_data.leftsY_gpu, leftsY, cm->mb_colsY * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_data.leftsUV_gpu, leftsUV, cm->mb_colsUV * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_data.rightsY_gpu, rightsY, cm->mb_colsY * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_data.rightsUV_gpu, rightsUV, cm->mb_colsUV * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_data.topsY_gpu, topsY, cm->mb_rowsY * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_data.topsUV_gpu, topsUV, cm->mb_rowsUV * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_data.bottomsY_gpu, bottomsY, cm->mb_rowsY * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_data.bottomsUV_gpu, bottomsUV, cm->mb_rowsUV * sizeof(int), hipMemcpyHostToDevice);

	cm->cuda_data.boundariesY.lefts = cm->cuda_data.leftsY_gpu;
	cm->cuda_data.boundariesY.rights = cm->cuda_data.rightsY_gpu;
	cm->cuda_data.boundariesY.tops = cm->cuda_data.topsY_gpu;
	cm->cuda_data.boundariesY.bottoms = cm->cuda_data.bottomsY_gpu;

	struct corner_data* corner_datasY = new struct corner_data[4];

	// Top left
	corner_datasY[0].mb_x = 0;
	corner_datasY[0].mb_y = 0;
	corner_datasY[0].left = 0;
	corner_datasY[0].top = 0;
	corner_datasY[0].right = ME_RANGE_Y;
	corner_datasY[0].bottom = ME_RANGE_Y;

	// Top right
	corner_datasY[1].mb_x = cm->mb_colsY - 1;
	corner_datasY[1].mb_y = 0;
	corner_datasY[1].left = cm->ypw - 8 - ME_RANGE_Y;
	corner_datasY[1].top = 0;
	corner_datasY[1].right = cm->ypw - 8;
	corner_datasY[1].bottom = ME_RANGE_Y;

	// Bottom left
	corner_datasY[2].mb_x = 0;
	corner_datasY[2].mb_y = cm->mb_rowsY - 1;
	corner_datasY[2].left = 0;
	corner_datasY[2].top = cm->yph - 8 - ME_RANGE_Y;
	corner_datasY[2].right = ME_RANGE_Y;
	corner_datasY[2].bottom = cm->yph - 8;

	// Bottom right
	corner_datasY[3].mb_x = cm->mb_colsY - 1;
	corner_datasY[3].mb_y = cm->mb_rowsY - 1;
	corner_datasY[3].left = cm->ypw - 8 - ME_RANGE_Y;
	corner_datasY[3].top = cm->yph - 8 - ME_RANGE_Y;
	corner_datasY[3].right = cm->ypw - 8;
	corner_datasY[3].bottom = cm->yph - 8;

	hipMemcpy(cm->cuda_data.corner_datasY_gpu, corner_datasY, 4 * sizeof(struct corner_data), hipMemcpyHostToDevice);

	delete[] corner_datasY;

	struct corner_data* semicornerH_datasY = new struct corner_data[8];

	// Top left2
	semicornerH_datasY[0].mb_x = 1;
	semicornerH_datasY[0].mb_y = 0;
	semicornerH_datasY[0].left = 0;
	semicornerH_datasY[0].top = 0;
	semicornerH_datasY[0].right = ME_RANGE_Y + 8;
	semicornerH_datasY[0].bottom = ME_RANGE_Y;

	// Top right2
	semicornerH_datasY[1].mb_x = cm->mb_colsY - 2;
	semicornerH_datasY[1].mb_y = 0;
	semicornerH_datasY[1].left = cm->ypw - 16 - ME_RANGE_Y;
	semicornerH_datasY[1].top = 0;
	semicornerH_datasY[1].right = cm->ypw - 8;
	semicornerH_datasY[1].bottom = ME_RANGE_Y;

	// Bottom left2
	semicornerH_datasY[2].mb_x = 1;
	semicornerH_datasY[2].mb_y = cm->mb_rowsY - 1;
	semicornerH_datasY[2].left = 0;
	semicornerH_datasY[2].top = cm->yph - 8 - ME_RANGE_Y;
	semicornerH_datasY[2].right = ME_RANGE_Y + 8;
	semicornerH_datasY[2].bottom = cm->yph - 8;

	// Bottom right2
	semicornerH_datasY[3].mb_x = cm->mb_colsY - 2;
	semicornerH_datasY[3].mb_y = cm->mb_rowsY - 1;
	semicornerH_datasY[3].left = cm->ypw - 16 - ME_RANGE_Y;
	semicornerH_datasY[3].top = cm->yph - 8 - ME_RANGE_Y;
	semicornerH_datasY[3].right = cm->ypw - 8;
	semicornerH_datasY[3].bottom = cm->yph - 8;

	hipMemcpy(cm->cuda_data.semicornerH_datasY_gpu, semicornerH_datasY, 8 * sizeof(struct corner_data), hipMemcpyHostToDevice);

	delete[] semicornerH_datasY;

	struct corner_data* semicornerV_datasY = new struct corner_data[8];

	// Top2 left
	semicornerV_datasY[0].mb_x = 0;
	semicornerV_datasY[0].mb_y = 1;
	semicornerV_datasY[0].left = 0;
	semicornerV_datasY[0].top = 0;
	semicornerV_datasY[0].right = ME_RANGE_Y;
	semicornerV_datasY[0].bottom = ME_RANGE_Y + 8;

	// Top2 right
	semicornerV_datasY[1].mb_x = cm->mb_colsY - 1;
	semicornerV_datasY[1].mb_y = 1;
	semicornerV_datasY[1].left = cm->ypw - 8 - ME_RANGE_Y;
	semicornerV_datasY[1].top = 0;
	semicornerV_datasY[1].right = cm->ypw - 8;
	semicornerV_datasY[1].bottom = ME_RANGE_Y + 8;

	// Bottom2 left
	semicornerV_datasY[2].mb_x = 0;
	semicornerV_datasY[2].mb_y = cm->mb_rowsY - 2;
	semicornerV_datasY[2].left = 0;
	semicornerV_datasY[2].top = cm->yph - 16 - ME_RANGE_Y;
	semicornerV_datasY[2].right = ME_RANGE_Y;
	semicornerV_datasY[2].bottom = cm->yph - 8;

	// Bottom2 right
	semicornerV_datasY[3].mb_x = cm->mb_colsY - 1;
	semicornerV_datasY[3].mb_y = cm->mb_rowsY - 2;
	semicornerV_datasY[3].left = cm->ypw - 8 - ME_RANGE_Y;
	semicornerV_datasY[3].top = cm->yph - 16 - ME_RANGE_Y;
	semicornerV_datasY[3].right = cm->ypw - 8;
	semicornerV_datasY[3].bottom = cm->yph - 8;

	hipMemcpy(cm->cuda_data.semicornerV_datasY_gpu, semicornerV_datasY, 8 * sizeof(struct corner_data), hipMemcpyHostToDevice);

	delete[] semicornerV_datasY;

	struct corner_data* innercorner_datasY = new struct corner_data[4];

	// Top left
	innercorner_datasY[0].mb_x = 1;
	innercorner_datasY[0].mb_y = 1;
	innercorner_datasY[0].left = 0;
	innercorner_datasY[0].top = 0;
	innercorner_datasY[0].right = ME_RANGE_Y + 8;
	innercorner_datasY[0].bottom = ME_RANGE_Y + 8;

	// Top right
	innercorner_datasY[1].mb_x = cm->mb_colsY - 2;
	innercorner_datasY[1].mb_y = 1;
	innercorner_datasY[1].left = cm->ypw - 16 - ME_RANGE_Y;
	innercorner_datasY[1].top = 0;
	innercorner_datasY[1].right = cm->ypw - 8;
	innercorner_datasY[1].bottom = ME_RANGE_Y + 8;

	int mb_x;
	int mb_y;

	// Bottom left
	mb_x = 1;
	mb_y = cm->mb_rowsY - 2;
	innercorner_datasY[2].mb_x = mb_x;
	innercorner_datasY[2].mb_y = mb_y;
	innercorner_datasY[2].left = leftsY[mb_x];
	innercorner_datasY[2].top = topsY[mb_y];
	innercorner_datasY[2].right = rightsY[mb_x];
	innercorner_datasY[2].bottom = bottomsY[mb_y];

	// Bottom right
	mb_x = cm->mb_colsY - 2;
	mb_y = cm->mb_rowsY - 2;
	innercorner_datasY[3].mb_x = mb_x;
	innercorner_datasY[3].mb_y = mb_y;
	innercorner_datasY[3].left = leftsY[mb_x];
	innercorner_datasY[3].top = topsY[mb_y];
	innercorner_datasY[3].right = rightsY[mb_x];
	innercorner_datasY[3].bottom = bottomsY[mb_y];

	hipMemcpy(cm->cuda_data.innercorner_datasY_gpu, innercorner_datasY, 4 * sizeof(struct corner_data), hipMemcpyHostToDevice);

	delete[] innercorner_datasY;

	delete[] leftsY;
	delete[] leftsUV;
	delete[] rightsY;
	delete[] rightsUV;
	delete[] topsY;
	delete[] topsUV;
	delete[] bottomsY;
	delete[] bottomsUV;
}

static void init_cuda_data(c63_common* cm)
{
	cuda_data* cuda_me = &(cm->cuda_data);

	hipStreamCreate(&cuda_me->streamY);
	hipStreamCreate(&cuda_me->streamU);
	hipStreamCreate(&cuda_me->streamV);

	hipMalloc((void**) &cuda_me->sad_index_resultsY, cm->mb_colsY*cm->mb_rowsY*sizeof(unsigned int));
	hipMalloc((void**) &cuda_me->sad_index_resultsU, cm->mb_colsUV*cm->mb_rowsUV*sizeof(unsigned int));
	hipMalloc((void**) &cuda_me->sad_index_resultsV, cm->mb_colsUV*cm->mb_rowsUV*sizeof(unsigned int));

	hipMalloc((void**) &(cuda_me->leftsY_gpu), cm->mb_colsY * sizeof(int));
	hipMalloc((void**) &(cuda_me->leftsUV_gpu), cm->mb_colsUV * sizeof(int));
	hipMalloc((void**) &(cuda_me->rightsY_gpu), cm->mb_colsY * sizeof(int));
	hipMalloc((void**) &(cuda_me->rightsUV_gpu), cm->mb_colsUV * sizeof(int));
	hipMalloc((void**) &(cuda_me->topsY_gpu), cm->mb_rowsY * sizeof(int));
	hipMalloc((void**) &(cuda_me->topsUV_gpu), cm->mb_rowsUV * sizeof(int));
	hipMalloc((void**) &(cuda_me->bottomsY_gpu), cm->mb_rowsY * sizeof(int));
	hipMalloc((void**) &(cuda_me->bottomsUV_gpu), cm->mb_rowsUV * sizeof(int));

	hipMalloc((void**) &(cuda_me->corner_datasY_gpu), 4 * sizeof(struct corner_data));
	hipMalloc((void**) &(cuda_me->innercorner_datasY_gpu), 4 * sizeof(struct corner_data));
	hipMalloc((void**) &(cuda_me->semicornerH_datasY_gpu), 8 * sizeof(struct corner_data));
	hipMalloc((void**) &(cuda_me->semicornerV_datasY_gpu), 8 * sizeof(struct corner_data));

	set_searchrange_boundaries_cuda(cm);
}

static void cleanup_cuda_data(c63_common* cm)
{
	hipStreamDestroy(cm->cuda_data.streamY);
	hipStreamDestroy(cm->cuda_data.streamU);
	hipStreamDestroy(cm->cuda_data.streamV);

	hipFree(cm->cuda_data.sad_index_resultsY);
	hipFree(cm->cuda_data.sad_index_resultsU);
	hipFree(cm->cuda_data.sad_index_resultsV);

	hipFree(cm->cuda_data.leftsY_gpu);
	hipFree(cm->cuda_data.leftsUV_gpu);
	hipFree(cm->cuda_data.rightsY_gpu);
	hipFree(cm->cuda_data.rightsUV_gpu);
	hipFree(cm->cuda_data.topsY_gpu);
	hipFree(cm->cuda_data.topsUV_gpu);
	hipFree(cm->cuda_data.bottomsY_gpu);
	hipFree(cm->cuda_data.bottomsUV_gpu);

	hipFree(cm->cuda_data.corner_datasY_gpu);
	hipFree(cm->cuda_data.innercorner_datasY_gpu);
	hipFree(cm->cuda_data.semicornerH_datasY_gpu);
	hipFree(cm->cuda_data.semicornerV_datasY_gpu);
}

static void copy_image_to_gpu(struct c63_common* cm, yuv_t* image, yuv_t* image_gpu)
{
	hipMemcpyAsync(image_gpu->Y, image->Y, cm->ypw * cm->yph * sizeof(uint8_t), hipMemcpyHostToDevice, cm->cuda_data.streamY);
	hipMemcpyAsync(image_gpu->U, image->U, cm->upw * cm->uph * sizeof(uint8_t), hipMemcpyHostToDevice, cm->cuda_data.streamU);
	hipMemcpyAsync(image_gpu->V, image->V, cm->vpw * cm->vph * sizeof(uint8_t), hipMemcpyHostToDevice, cm->cuda_data.streamV);
}

struct c63_common* init_c63_enc(int width, int height)
{
  int i;

  /* calloc() sets allocated memory to zero */
  struct c63_common *cm = (c63_common*) calloc(1, sizeof(struct c63_common));

  cm->width = width;
  cm->height = height;

  cm->padw[Y_COMPONENT] = cm->ypw = (uint32_t)(ceil(width/16.0f)*16);
  cm->padh[Y_COMPONENT] = cm->yph = (uint32_t)(ceil(height/16.0f)*16);
  cm->padw[U_COMPONENT] = cm->upw = (uint32_t)(ceil(width*UX/(YX*8.0f))*8);
  cm->padh[U_COMPONENT] = cm->uph = (uint32_t)(ceil(height*UY/(YY*8.0f))*8);
  cm->padw[V_COMPONENT] = cm->vpw = (uint32_t)(ceil(width*VX/(YX*8.0f))*8);
  cm->padh[V_COMPONENT] = cm->vph = (uint32_t)(ceil(height*VY/(YY*8.0f))*8);

  cm->mb_colsY = cm->ypw / 8;
  cm->mb_rowsY = cm->yph / 8;
  cm->mb_colsUV = cm->mb_colsY / 2;
  cm->mb_rowsUV = cm->mb_rowsY / 2;

  /* Quality parameters -- Home exam deliveries should have original values,
   i.e., quantization factor should be 25, search range should be 16, and the
   keyframe interval should be 100. */
  cm->qp = 25;                  // Constant quantization factor. Range: [1..50]
  //cm->me_search_range = 16;   // This is now defined in c63.h
  cm->keyframe_interval = 100;  // Distance between keyframes

  /* Initialize quantization tables */
  for (i = 0; i < 64; ++i)
  {
    cm->quanttbl[Y_COMPONENT][i] = yquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[U_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[V_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
  }

  cm->curframe = create_frame(cm);
  cm->refframe = create_frame(cm);

  init_cuda_data(cm);

  return cm;
}

void free_c63_enc(struct c63_common* cm)
{
	destroy_frame(cm->curframe);
	destroy_frame(cm->refframe);
	free(cm);
}

static void print_help()
{
  printf("Usage: ./c63enc [options] input_file\n");
  printf("Commandline options:\n");
  printf("  -h                             Height of images to compress\n");
  printf("  -w                             Width of images to compress\n");
  printf("  -o                             Output file (.c63)\n");
  printf("  [-f]                           Limit number of frames to encode\n");
  printf("\n");

  exit(EXIT_FAILURE);
}

int main(int argc, char **argv)
{
	int c;

	if (argc == 1) { print_help(); }

	while ((c = getopt(argc, argv, "h:w:o:f:i:")) != -1)
	{
		switch (c)
		{
		case 'h':
			height = atoi(optarg);
			break;
		case 'w':
			width = atoi(optarg);
			break;
		case 'o':
			output_file = optarg;
			break;
		case 'f':
			limit_numframes = atoi(optarg);
			break;
		default:
			print_help();
			break;
		}
	}

	if (optind >= argc)
	{
		fprintf(stderr, "Error getting program options, try --help.\n");
		exit(EXIT_FAILURE);
	}

	outfile = fopen(output_file, "wb");

	if (outfile == NULL)
	{
		perror("fopen");
		exit(EXIT_FAILURE);
	}

	struct c63_common *cm = init_c63_enc(width, height);
	cm->e_ctx.fp = outfile;

	struct c63_common *cm2 = init_c63_enc(width, height);
	cm2->e_ctx.fp = outfile;

	input_file = argv[optind];

	if (limit_numframes) { printf("Limited to %d frames.\n", limit_numframes); }

	FILE *infile = fopen(input_file, "rb");

	if (infile == NULL)
	{
		perror("fopen");
		exit(EXIT_FAILURE);
	}

	/* Encode input frames */
	int numframes = 0;

	yuv_t *image = create_image(cm);
	yuv_t *image_gpu = create_image_gpu(cm);

	// Read the first image (image 0) from disk
	bool ok = read_yuv(infile, image);

	if (ok) {
		// Copy the first image to GPU asynchronously
		copy_image_to_gpu(cm, image, image_gpu);

		printf("Encoding frame %d, ", numframes);
		++numframes;

		// Start encoding the first image asynchronously
		c63_encode_image(cm, image_gpu);
		++cm->framenum;
		++cm->frames_since_keyframe;
		++cm2->framenum;
		++cm2->frames_since_keyframe;

		while (!limit_numframes || numframes < limit_numframes)
		{
			// Read the current image from disk
			ok = read_yuv(infile, image);
			if (!ok)
			{
				break;
			}

			// We need the reconstructed previous image
			std::swap(cm->curframe->recons_gpu, cm2->curframe->recons_gpu);

			// Wait until the previous image has been encoded
			hipStreamSynchronize(cm->cuda_data.streamY);
			hipStreamSynchronize(cm->cuda_data.streamU);
			hipStreamSynchronize(cm->cuda_data.streamV);
			printf("Done!\n");

			// Copy the current image to GPU asynchronously
			copy_image_to_gpu(cm2, image, image_gpu);

			printf("Encoding frame %d, ", numframes);
			++numframes;

			// Start encoding the current image asynchronously
			c63_encode_image(cm2, image_gpu);
			++cm->framenum;
			++cm->frames_since_keyframe;
			++cm2->framenum;
			++cm2->frames_since_keyframe;

			// While the GPU is busy, we can write the previous frame to disk
			write_frame(cm);

			// Swap the pointers so we can use this loop for even and odd numbered images
			std::swap(cm, cm2);
		}

		// Wait until the last image has been encoded
		hipStreamSynchronize(cm->cuda_data.streamY);
		hipStreamSynchronize(cm->cuda_data.streamU);
		hipStreamSynchronize(cm->cuda_data.streamV);
		printf("Done!\n");

		// Write the last frame to disk
		write_frame(cm);
	}

	destroy_image(image);
	destroy_image_gpu(image_gpu);

	cleanup_cuda_data(cm);
	free_c63_enc(cm);

	cleanup_cuda_data(cm2);
	free_c63_enc(cm2);

	fclose(outfile);
	fclose(infile);

	return EXIT_SUCCESS;
}
