#include "hip/hip_runtime.h"
extern "C" {
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "c63.h"
#include "c63_write.h"
#include "tables.h"
}

#include "common.h"
#include "dsp.h"
#include "me.h"

static char *output_file, *input_file;
FILE *outfile;

static int limit_numframes = 0;

static uint32_t width;
static uint32_t height;

/* getopt */
extern int optind;
extern char *optarg;

// Temporary buffers
int16_t *gpu_Y_16;
int16_t *gpu_U_16;
int16_t *gpu_V_16;

// Get CPU cycle count
uint64_t rdtsc(){
    unsigned int lo,hi;
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return ((uint64_t)hi << 32) | lo;
}

/* Read planar YUV frames with 4:2:0 chroma sub-sampling */
static bool read_yuv(FILE *file, struct c63_common *cm, yuv_t* image)
{
  size_t len = 0;

  /* Read Y. The size of Y is the same as the size of the image. The indices
     represents the color component (0 is Y, 1 is U, and 2 is V) */
  len += fread(image->Y, 1, width*height, file);

  /* Read U. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y
     because (height/2)*(width/2) = (height*width)/4. */
  len += fread(image->U, 1, (width*height)/4, file);

  /* Read V. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y. */
  len += fread(image->V, 1, (width*height)/4, file);

  if (ferror(file))
  {
    perror("ferror");
    exit(EXIT_FAILURE);
  }

  if (feof(file))
  {
    return false;
  }
  else if (len != width*height*1.5)
  {
    fprintf(stderr, "Reached end of file, but incorrect bytes read.\n");
    fprintf(stderr, "Wrong input? (height: %d width: %d)\n", height, width);

    return false;
  }

  return true;
}

static void zero_out_prediction(struct c63_common* cm)
{
	struct frame* frame = cm->curframe;
	memset(frame->predicted->Y, 0, cm->ypw * cm->yph * sizeof(uint8_t));
	memset(frame->predicted->U, 0, cm->upw * cm->uph * sizeof(uint8_t));
	memset(frame->predicted->V, 0, cm->vpw * cm->vph * sizeof(uint8_t));

	hipMemset(frame->predicted_gpu->Y, 0, cm->ypw * cm->yph * sizeof(uint8_t));
	hipMemset(frame->predicted_gpu->U, 0, cm->upw * cm->uph * sizeof(uint8_t));
	hipMemset(frame->predicted_gpu->V, 0, cm->vpw * cm->vph * sizeof(uint8_t));
}

static void c63_encode_image(struct c63_common *cm, yuv_t *image, yuv_t* image_gpu)
{
	// Advance to next frame by swapping current and reference frame
	struct frame* temp = cm->refframe;
	cm->refframe = cm->curframe;
	cm->curframe = temp;

	cm->curframe->orig = image;
	cm->curframe->orig_gpu = image_gpu;

	/* Check if keyframe */
	if (cm->framenum == 0 || cm->frames_since_keyframe == cm->keyframe_interval)
	{
		cm->curframe->keyframe = 1;
		cm->frames_since_keyframe = 0;

		fprintf(stderr, " (keyframe) ");
	}
	else { cm->curframe->keyframe = 0; }

	if (!cm->curframe->keyframe)
	{
		/* Motion Estimation */
		c63_motion_estimate(cm);

		/* Motion Compensation */
		c63_motion_compensate(cm);
	}
	else
	{
		// dct_quantize() expects zeroed out prediction buffers for key frames.
		// We zero them out here since we reuse the buffers from previous frames.
		zero_out_prediction(cm);
	}

	yuv_t* predicted = cm->curframe->predicted_gpu;

	/* DCT and Quantization */
	dct_quantize(cm->curframe->orig_gpu->Y, predicted->Y, cm->padw[Y_COMPONENT],
			cm->padh[Y_COMPONENT], gpu_Y_16, cm->curframe->residuals->Ydct,
			Y_COMPONENT);

	dct_quantize(cm->curframe->orig_gpu->U, predicted->U, cm->padw[U_COMPONENT],
			cm->padh[U_COMPONENT], gpu_U_16, cm->curframe->residuals->Udct,
			U_COMPONENT);

	dct_quantize(cm->curframe->orig_gpu->V, predicted->V, cm->padw[V_COMPONENT],
			cm->padh[V_COMPONENT], gpu_V_16, cm->curframe->residuals->Vdct,
			V_COMPONENT);

	/* Reconstruct frame for inter-prediction */
	dequantize_idct(gpu_Y_16, predicted->Y,
			cm->ypw, cm->yph, cm->curframe->recons_gpu->Y, cm->curframe->recons->Y, Y_COMPONENT);
	dequantize_idct(gpu_U_16, predicted->U,
			cm->upw, cm->uph, cm->curframe->recons_gpu->U, cm->curframe->recons->U, U_COMPONENT);
	dequantize_idct(gpu_V_16, predicted->V,
			cm->vpw, cm->vph, cm->curframe->recons_gpu->V, cm->curframe->recons->V, V_COMPONENT);

	/* Function dump_image(), found in common.c, can be used here to check if the
     prediction is correct */

	write_frame(cm);

	++cm->framenum;
	++cm->frames_since_keyframe;
}

static void set_searchrange_boundaries_cuda(c63_common* cm)
{
	int hY = cm->padh[Y_COMPONENT];
	int hUV = cm->padh[U_COMPONENT];

	int wY = cm->padw[Y_COMPONENT];
	int wUV = cm->padw[U_COMPONENT];

	int* leftsY = new int[cm->mb_colsY];
	int* leftsUV = new int[cm->mb_colsUV];
	int* rightsY = new int[cm->mb_colsY];
	int* rightsUV = new int[cm->mb_colsUV];
	int* topsY = new int[cm->mb_rowsY];
	int* topsUV = new int[cm->mb_rowsUV];
	int* bottomsY = new int[cm->mb_rowsY];
	int* bottomsUV = new int[cm->mb_rowsUV];

	for (int mb_x = 0; mb_x < cm->mb_colsY; ++mb_x) {
		leftsY[mb_x] = mb_x * 8 - ME_RANGE_Y;
		rightsY[mb_x] = mb_x * 8 + ME_RANGE_Y;

		if (leftsY[mb_x] < 0) {
			leftsY[mb_x] = 0;
		}

		if (rightsY[mb_x] > (wY - 8)) {
			rightsY[mb_x] = wY - 8;
		}
	}

	for (int mb_x = 0; mb_x < cm->mb_colsUV; ++mb_x) {
		leftsUV[mb_x] = mb_x * 8 - ME_RANGE_UV;
		rightsUV[mb_x] = mb_x * 8 + ME_RANGE_UV;

		if (leftsUV[mb_x] < 0) {
			leftsUV[mb_x] = 0;
		}

		if (rightsUV[mb_x] > (wUV - 8)) {
			rightsUV[mb_x] = wUV - 8;
		}
	}

	for (int mb_y = 0; mb_y < cm->mb_rowsY; ++mb_y) {
		topsY[mb_y] = mb_y * 8 - ME_RANGE_Y;
		bottomsY[mb_y] = mb_y * 8 + ME_RANGE_Y;

		if (topsY[mb_y] < 0) {
			topsY[mb_y] = 0;
		}

		if (bottomsY[mb_y] > (hY - 8)) {
			bottomsY[mb_y] = hY - 8;
		}
	}

	for (int mb_y = 0; mb_y < cm->mb_rowsUV; ++mb_y) {
		topsUV[mb_y] = mb_y * 8 - ME_RANGE_UV;
		bottomsUV[mb_y] = mb_y * 8 + ME_RANGE_UV;

		if (topsUV[mb_y] < 0) {
			topsUV[mb_y] = 0;
		}

		if (bottomsUV[mb_y] > (hUV - 8)) {
			bottomsUV[mb_y] = hUV - 8;
		}
	}

	hipMemcpy(cm->cuda_me.leftsY_gpu, leftsY, cm->mb_colsY * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.leftsUV_gpu, leftsUV, cm->mb_colsUV * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.rightsY_gpu, rightsY, cm->mb_colsY * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.rightsUV_gpu, rightsUV, cm->mb_colsUV * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.topsY_gpu, topsY, cm->mb_rowsY * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.topsUV_gpu, topsUV, cm->mb_rowsUV * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.bottomsY_gpu, bottomsY, cm->mb_rowsY * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cm->cuda_me.bottomsUV_gpu, bottomsUV, cm->mb_rowsUV * sizeof(int), hipMemcpyHostToDevice);

	delete[] leftsY;
	delete[] leftsUV;
	delete[] rightsY;
	delete[] rightsUV;
	delete[] topsY;
	delete[] topsUV;
	delete[] bottomsY;
	delete[] bottomsUV;
}

static void init_cuda_data(c63_common* cm)
{
	cuda_data_me* cuda_me = &(cm->cuda_me);

	hipMalloc((void**) &(cuda_me->leftsY_gpu), cm->mb_colsY * sizeof(int));
	hipMalloc((void**) &(cuda_me->leftsUV_gpu), cm->mb_colsUV * sizeof(int));
	hipMalloc((void**) &(cuda_me->rightsY_gpu), cm->mb_colsY * sizeof(int));
	hipMalloc((void**) &(cuda_me->rightsUV_gpu), cm->mb_colsUV * sizeof(int));
	hipMalloc((void**) &(cuda_me->topsY_gpu), cm->mb_rowsY * sizeof(int));
	hipMalloc((void**) &(cuda_me->topsUV_gpu), cm->mb_rowsUV * sizeof(int));
	hipMalloc((void**) &(cuda_me->bottomsY_gpu), cm->mb_rowsY * sizeof(int));
	hipMalloc((void**) &(cuda_me->bottomsUV_gpu), cm->mb_rowsUV * sizeof(int));

	hipMalloc(&gpu_Y_16, cm->padw[Y_COMPONENT]*cm->padh[Y_COMPONENT]*sizeof(int16_t));
	hipMalloc(&gpu_U_16, cm->padw[U_COMPONENT]*cm->padh[U_COMPONENT]*sizeof(int16_t));
	hipMalloc(&gpu_V_16, cm->padw[V_COMPONENT]*cm->padh[V_COMPONENT]*sizeof(int16_t));

	set_searchrange_boundaries_cuda(cm);
}

static void cleanup_cuda_data(c63_common* cm)
{
	hipFree(cm->cuda_me.leftsY_gpu);
	hipFree(cm->cuda_me.leftsUV_gpu);
	hipFree(cm->cuda_me.rightsY_gpu);
	hipFree(cm->cuda_me.rightsUV_gpu);
	hipFree(cm->cuda_me.topsY_gpu);
	hipFree(cm->cuda_me.topsUV_gpu);
	hipFree(cm->cuda_me.bottomsY_gpu);
	hipFree(cm->cuda_me.bottomsUV_gpu);

	hipFree(gpu_Y_16);
	hipFree(gpu_U_16);
	hipFree(gpu_V_16);
}

static void copy_image_to_gpu(struct c63_common* cm, yuv_t* image, yuv_t* image_gpu)
{
	hipMemcpy(image_gpu->Y, image->Y, cm->ypw * cm->yph * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(image_gpu->U, image->U, cm->upw * cm->uph * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(image_gpu->V, image->V, cm->vpw * cm->vph * sizeof(uint8_t), hipMemcpyHostToDevice);
}

struct c63_common* init_c63_enc(int width, int height)
{
  int i;

  /* calloc() sets allocated memory to zero */
  struct c63_common *cm = (c63_common*) calloc(1, sizeof(struct c63_common));

  cm->width = width;
  cm->height = height;

  cm->padw[Y_COMPONENT] = cm->ypw = (uint32_t)(ceil(width/16.0f)*16);
  cm->padh[Y_COMPONENT] = cm->yph = (uint32_t)(ceil(height/16.0f)*16);
  cm->padw[U_COMPONENT] = cm->upw = (uint32_t)(ceil(width*UX/(YX*8.0f))*8);
  cm->padh[U_COMPONENT] = cm->uph = (uint32_t)(ceil(height*UY/(YY*8.0f))*8);
  cm->padw[V_COMPONENT] = cm->vpw = (uint32_t)(ceil(width*VX/(YX*8.0f))*8);
  cm->padh[V_COMPONENT] = cm->vph = (uint32_t)(ceil(height*VY/(YY*8.0f))*8);

  cm->mb_colsY = cm->ypw / 8;
  cm->mb_rowsY = cm->yph / 8;
  cm->mb_colsUV = cm->mb_colsY / 2;
  cm->mb_rowsUV = cm->mb_rowsY / 2;

  /* Quality parameters -- Home exam deliveries should have original values,
   i.e., quantization factor should be 25, search range should be 16, and the
   keyframe interval should be 100. */
  cm->qp = 25;                  // Constant quantization factor. Range: [1..50]
  //cm->me_search_range = 16;   // This is now defined in c63.h
  cm->keyframe_interval = 100;  // Distance between keyframes

  /* Initialize quantization tables */
  for (i = 0; i < 64; ++i)
  {
    cm->quanttbl[Y_COMPONENT][i] = yquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[U_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
    cm->quanttbl[V_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
  }

  cm->curframe = create_frame(cm);
  cm->refframe = create_frame(cm);

  init_cuda_data(cm);

  return cm;
}

void free_c63_enc(struct c63_common* cm)
{
	destroy_frame(cm->curframe);
	destroy_frame(cm->refframe);
	free(cm);
}

static void print_help()
{
  printf("Usage: ./c63enc [options] input_file\n");
  printf("Commandline options:\n");
  printf("  -h                             Height of images to compress\n");
  printf("  -w                             Width of images to compress\n");
  printf("  -o                             Output file (.c63)\n");
  printf("  [-f]                           Limit number of frames to encode\n");
  printf("\n");

  exit(EXIT_FAILURE);
}

int main(int argc, char **argv)
{
	int c;

	if (argc == 1) { print_help(); }

	while ((c = getopt(argc, argv, "h:w:o:f:i:")) != -1)
	{
		switch (c)
		{
		case 'h':
			height = atoi(optarg);
			break;
		case 'w':
			width = atoi(optarg);
			break;
		case 'o':
			output_file = optarg;
			break;
		case 'f':
			limit_numframes = atoi(optarg);
			break;
		default:
			print_help();
			break;
		}
	}

	if (optind >= argc)
	{
		fprintf(stderr, "Error getting program options, try --help.\n");
		exit(EXIT_FAILURE);
	}

	outfile = fopen(output_file, "wb");

	if (outfile == NULL)
	{
		perror("fopen");
		exit(EXIT_FAILURE);
	}

	struct c63_common *cm = init_c63_enc(width, height);
	cm->e_ctx.fp = outfile;

	input_file = argv[optind];

	if (limit_numframes) { printf("Limited to %d frames.\n", limit_numframes); }

	FILE *infile = fopen(input_file, "rb");

	if (infile == NULL)
	{
		perror("fopen");
		exit(EXIT_FAILURE);
	}

	/* Encode input frames */
	int numframes = 0;

	# ifdef SHOW_CYCLES
	uint64_t kCycleCountTotal = 0;
	# endif

	yuv_t *image = create_image(cm);
	yuv_t *image_gpu = create_image_gpu(cm);

	while (1)
	{
		bool ok = read_yuv(infile, cm, image);

		if (!ok) { break; }

		copy_image_to_gpu(cm, image, image_gpu);

		printf("Encoding frame %d, ", numframes);

	# ifdef SHOW_CYCLES
		uint64_t cycleCountBefore = rdtsc();
		c63_encode_image(cm, image, image_gpu);
		uint64_t cycleCountAfter = rdtsc();

		uint64_t kCycleCount = (cycleCountAfter - cycleCountBefore)/1000;
		kCycleCountTotal += kCycleCount;
		printf("%" PRIu64 "k cycles, ", kCycleCount);
	# else
		c63_encode_image(cm, image, image_gpu);
	# endif

		printf("Done!\n");

		++numframes;

		if (limit_numframes && numframes >= limit_numframes) { break; }
	}

# ifdef SHOW_CYCLES
	printf("-----------\n");
	printf("Average CPU cycle count per frame: %" PRIu64 "k\n", kCycleCountTotal/numframes);
# endif

	destroy_image(image);
	destroy_image_gpu(image_gpu);
	cleanup_cuda_data(cm);

	free_c63_enc(cm);
	fclose(outfile);
	fclose(infile);

	return EXIT_SUCCESS;
}
